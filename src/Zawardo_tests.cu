#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief The start of Zawarudo
///

#include <hip/hip_runtime.h>
#include <mkl.h>
#include "hipblas.h"

#include "cpu_utils.hpp"
#include "gpu_utils.hpp"

/// Inputs: A, B, C, alpha, beta, M, N, K, store format (Colmajor OR Rowmajor)
/// for A, B, C
/// TODO: Add lda, ldb, ldc of initial call in order to also predict cases for
/// sub-matrix gemm ( if ld_dim > dim etc)

/// Predictor inputs: M_split (OR) N_split (OR) K_split, <<cblas Layout (for
/// CPU), >>, C_add device
/// TODO: Also add asynch trans

/// Output: A, B intact, C dgemm result
/// TODO: No mem leaks (CPU and GPU)

/// Extras: Precise debug and parameter error checking
/// TODO: More than two devices

enum mem_layout { ROW_MAJOR = 0, COL_MAJOR };

const char *print_mem(mem_layout mem) {
  if (mem == ROW_MAJOR)
    return "Row major";
  else if (mem == COL_MAJOR)
    return "Col major";
  else
    return "ERROR";
}

typedef struct control_str {
  // For now assume device 0 is always the host
  size_t num_devices;
  mem_layout A_mem;
  mem_layout B_mem;
  mem_layout C_mem;
  double pin_alloc_t = 0, scatter_t = 0, transpose_t = 0, cpu_ex_t = 0,
         gpu_ex_t = 0, gather_t = 0, reduce_t = 0;

} * control_p;

typedef struct pred_str {
  size_t M_split;
  size_t N_split;
  size_t K_split;
  CBLAS_LAYOUT cblas_target;
  size_t Cadd_device_id;
  // TODO: Not implemented
  int asynch_trans = 0;

} * predict_p;

double *gpu_dgemm(control_p ctrl, predict_p pred, size_t M, size_t N, size_t K,
                  double alpha, double *A, double *B, double beta, double *C) {
  debug("-> gpu_dgemm()");
  if (ctrl->num_devices < 1)
    error(
        "gpu_dgemm -> 0 or less devices? What are you trying to "
        "do...");
  else if (ctrl->num_devices > 2)
    error(
        "gpu_dgemm -> Max 1 GPU + 1 CPU implemented (nice "
        "try,though).");

  if (pred->asynch_trans)
    error(
        "gpu_dgemm -> asynch transactions not implemented yet "
        "(nice try,though).");

  if (!A) error("gpu_dgemm -> A is not malloc'ed correctly");
  if (!B) error("gpu_dgemm -> B is not malloc'ed correctly");
  if (beta != 0 && !C) error("gpu_dgemm -> C is not malloc'ed correctly");
  debug(
      "gpu_dgemm -> Trying your Matrix bounds (incomming "
      "segfaults)...");
  double test = A[M * K - 1];
  test = B[K * N - 1];
  if (beta != 0) test = C[M * N - 1];
  debug("gpu_dgemm -> Passed.");
  double *C_out, local_t;

  double *A_gpu, *B_gpu, *C_gpu;

  double *C_T, cpu_beta = 0, gpu_beta = 0;

  size_t M_gpu = pred->M_split, ldA = 0, ldB = 0, ldC = 0, d_ldA = 0, d_ldB = 0,
         d_ldC = 0;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans

  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);

  gpu_timer_p cuda_timer = gpu_timer_init();

  if (ctrl->A_mem == COL_MAJOR)
    ldA = M;
  else
    ldA = K;
  if (ctrl->B_mem == COL_MAJOR)
    ldB = K;
  else
    ldB = N;

  if (ctrl->C_mem == COL_MAJOR) {
    ldC = M;
    if (ctrl->A_mem == COL_MAJOR)
      gpu_op_A = HIPBLAS_OP_N;
    else
      gpu_op_A = HIPBLAS_OP_T;
    if (ctrl->B_mem == COL_MAJOR)
      gpu_op_B = HIPBLAS_OP_N;
    else
      gpu_op_B = HIPBLAS_OP_T;
  } else {
    ldC = N;
    if (ctrl->A_mem == ROW_MAJOR)
      gpu_op_A = HIPBLAS_OP_T;
    else
      gpu_op_A = HIPBLAS_OP_N;

    if (ctrl->B_mem == ROW_MAJOR)
      gpu_op_B = HIPBLAS_OP_T;
    else
      gpu_op_B = HIPBLAS_OP_N;
  }

  local_t = csecond();
  A_gpu = Dvec_transfer_gpu(A, M * K);
  B_gpu = Dvec_transfer_gpu(B, K * N);
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  if (pred->Cadd_device_id == -1) {
    gpu_beta = beta;
    local_t = csecond();
    if (ctrl->C_mem == ROW_MAJOR) {
      C_T = (double *)pin_malloc(M * N * sizeof(double));
      Dtranspose(C_T, C, M, N);
    } else
      C_T = C;
    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;

    local_t = csecond();
    C_gpu = Dvec_transfer_gpu(C_T, M * N);
    local_t = csecond() - local_t;
    ctrl->scatter_t += local_t;
  } else if (pred->Cadd_device_id == 0) {
    local_t = csecond();
    C_gpu = (double *)gpu_malloc(M * N * sizeof(double));
    C_T = (double *)pin_malloc(M * N * sizeof(double));
    local_t = csecond() - local_t;
    ctrl->pin_alloc_t += local_t;
  }

  else if (pred->Cadd_device_id == 1) {
    debug(
        "gpu_dgemm -> pred->Cadd_device_id == 1 is obsolete..all computations "
        "on gpu anyway");

    gpu_beta = beta;
    local_t = csecond();
    if (ctrl->C_mem == ROW_MAJOR) {
      C_T = (double *)pin_malloc(M * N * sizeof(double));
      Dtranspose(C_T, C, M, N);
    } else
      C_T = C;
    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;

    local_t = csecond();
    C_gpu = Dvec_transfer_gpu(C_T, M * N);
    local_t = csecond() - local_t;
    ctrl->scatter_t += local_t;
  }

  gpu_timer_start(cuda_timer);
  stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M, N, K, &alpha, A_gpu, ldA,
                     B_gpu, ldB, &gpu_beta, C_gpu, M);

  gpu_timer_stop(cuda_timer);

  if (pred->Cadd_device_id == 0) cblas_dscal(M * N, beta, C, 1);

  cudaCheckErrors();
  ctrl->gpu_ex_t = gpu_timer_get(cuda_timer) / 1000;

  local_t = csecond();
  hipMemcpy(C_T, C_gpu, M * N * sizeof(double), hipMemcpyDeviceToHost);
  local_t = csecond() - local_t;
  ctrl->gather_t += local_t;

  if (ctrl->C_mem == ROW_MAJOR) {
    local_t = csecond();
    if (pred->Cadd_device_id == 0)
      Dtranspose_add(C, C_T, N, M);
    else
      Dtranspose(C, C_T, N, M);
    pin_free(C_T);
    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;
  } else {
    local_t = csecond();
    if (pred->Cadd_device_id == 0)
      cblas_daxpy(N * M, 1.0, C_T, 1, C, 1);
    else
      C = C_T;

    local_t = csecond() - local_t;
    ctrl->reduce_t += local_t;
  }

  printf(
      "Device overhead(M=%d, N=%d, K=%d) pin_alloc = %lf ms, scatter = %lf "
      "ms, "
      "transpose = %lf ms, gather = %lf ms, reduce = %lf ms\n",
      M, N, K, 1000 * ctrl->pin_alloc_t, 1000 * ctrl->scatter_t,
      1000 * ctrl->transpose_t, 1000 * ctrl->gather_t, 1000 * ctrl->reduce_t);

  printf("Hybrid Sgemm(M=%d, N=%d, K=%d) GPU time = ", M, N, K);
  report_results(ctrl->gpu_ex_t, (long)M * K * (2 * N + 1),
                 (long)(M * K + K * N + M * N * 2) *
                     sizeof(double));  //(M*N+(long)M*K*(3*N+1))
  printf("\n");
  // pin_free(C_buffer);

  // FIXME: implement correct frees with switches
  gpu_free(A_gpu);
  gpu_free(B_gpu);
  gpu_free(C_gpu);

  C_out = C;
  debug("<- gpu_dgemm()\n");
  return C_out;
}

double *hybrid_dgemm_Msplit(control_p ctrl, predict_p pred, size_t M, size_t N,
                            size_t K, double alpha, double *A, double *B,
                            double beta, double *C) {
  debug("-> hybrid_dgemm_Msplit()");
  if (ctrl->num_devices < 1)
    error(
        "hybrid_dgemm_Msplit -> 0 or less devices? What are you trying to "
        "do...");
  else if (ctrl->num_devices > 2)
    error(
        "hybrid_dgemm_Msplit -> Max 1 GPU + 1 CPU implemented (nice "
        "try,though).");

  if (pred->asynch_trans)
    error(
        "hybrid_dgemm_Msplit -> asynch transactions not implemented yet "
        "(nice try,though).");

  if (!pred->M_split || pred->M_split >= M)
    error("hybrid_dgemm_Msplit -> Full CPU/GPU versions do not belong here");
  if (!A) error("hybrid_dgemm_Msplit -> A is not malloc'ed correctly");
  if (!B) error("hybrid_dgemm_Msplit -> B is not malloc'ed correctly");
  if (beta != 0 && !C)
    error("hybrid_dgemm_Msplit -> C is not malloc'ed correctly");
  debug(
      "hybrid_dgemm_Msplit -> Trying your Matrix bounds (incomming "
      "segfaults)...");
  double test = A[M * K - 1];
  test = B[K * N - 1];
  if (beta != 0) test = C[M * N - 1];
  debug("hybrid_dgemm_Msplit -> Passed.");
  double *C_out, local_t;

  double *A_cpu, *A_gpu, *B_cpu, *B_gpu, *C_cpu, *C_gpu;

  double *A_T, *B_T, *C_T, *C_buffer, cpu_beta = 0, gpu_beta = 0;

  size_t M_gpu = pred->M_split, M_cpu = M - pred->M_split, ldA = 0, ldB = 0,
         ldC = 0, d_ldA = 0, d_ldB = 0, d_ldC = 0;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans

  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);

  gpu_timer_p cuda_timer = gpu_timer_init();

  /// Setup A parts on host and device

  switch (ctrl->A_mem) {
    case (ROW_MAJOR):
      local_t = csecond();
      A_gpu = Dvec_transfer_gpu(A, M_gpu * K);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
      gpu_op_A = HIPBLAS_OP_T;
      d_ldA = K;
      A_cpu = &(A[M_gpu * K]);
      ldA = K;
      if (pred->cblas_target == CblasRowMajor)
        cpu_op_A = CblasNoTrans;
      else if (pred->cblas_target == CblasColMajor)
        cpu_op_A = CblasTrans;
      break;
    case (COL_MAJOR):
      gpu_op_A = HIPBLAS_OP_N;
      d_ldA = M_gpu;

      if (pred->cblas_target == CblasRowMajor)
        error(
            "hybrid_dgemm_Msplit -> pred->cblas_target == CblasRowMajor and "
            "pred->A_mem == COL_MAJOR -> Mode not allowed.");
      else if (pred->cblas_target == CblasColMajor) {
        A_cpu = &(A[M_gpu]);
        ldA = M;
        cpu_op_A = CblasNoTrans;
      }
      local_t = csecond();
      A_gpu = Dvec_chunk_transfer_gpu(A, K, M_gpu, M);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;

      break;
    default:
      error("hybrid_dgemm_Msplit -> Unreachable default reached ");
  }

  /// Setup B parts on host and device
  B_cpu = B;
  local_t = csecond();
  B_gpu = Dvec_transfer_gpu(B, K * N);
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  switch (ctrl->B_mem) {
    case (ROW_MAJOR):
      gpu_op_B = HIPBLAS_OP_T;
      d_ldB = N;
      ldB = N;
      if (pred->cblas_target == CblasRowMajor)
        cpu_op_B = CblasNoTrans;
      else if (pred->cblas_target == CblasColMajor)
        cpu_op_B = CblasTrans;
      break;
    case (COL_MAJOR):
      gpu_op_B = HIPBLAS_OP_N;
      d_ldB = K;
      ldB = K;
      if (pred->cblas_target == CblasRowMajor)
        cpu_op_B = CblasTrans;
      else if (pred->cblas_target == CblasColMajor)
        cpu_op_B = CblasNoTrans;
      break;
    default:
      error("hybrid_dgemm_Msplit -> Unreachable default reached ");
  }

  /// Setup C parts on host and device
  d_ldC = M_gpu;

  if (!beta) {
    local_t = csecond();
    C_gpu = (double *)gpu_malloc(M_gpu * N * sizeof(double));
    C_buffer = (double *)pin_malloc(M_gpu * N * sizeof(double));
    if (!C)
      // TODO: Can we use only as much space as required?
      error("Unallocated input C not implemented");
    local_t = csecond() - local_t;
    ctrl->pin_alloc_t += local_t;

    if (ctrl->C_mem == ROW_MAJOR) {
      if (pred->cblas_target == CblasRowMajor) {
        C_cpu = &(C[M_gpu * N]);
        ldC = N;
      } else if (pred->cblas_target == CblasColMajor) {
        local_t = csecond();
        C_cpu = (double *)pin_malloc(M_cpu * N * sizeof(double));
        local_t = csecond() - local_t;
        ctrl->pin_alloc_t += local_t;
        ldC = M_cpu;
      }
    } else if (ctrl->C_mem == COL_MAJOR) {
      C_cpu = &(C[M_gpu]);
      if (pred->cblas_target == CblasRowMajor)
        error(
            "hybrid_dgemm_Msplit -> pred->cblas_target == CblasRowMajor and "
            "pred->C_mem == COL_MAJOR -> Mode not allowed.");
      else if (pred->cblas_target == CblasColMajor)
        ldC = M;
    }

  } else if (pred->Cadd_device_id == -1) {
    cpu_beta = gpu_beta = beta;
    if (ctrl->C_mem == ROW_MAJOR) {
      C_cpu = &(C[M_gpu * N]);
      if (pred->cblas_target == CblasRowMajor)
        ldC = N;
      else if (pred->cblas_target == CblasColMajor) {
        local_t = csecond();
        C_cpu = (double *)pin_malloc(M_cpu * N * sizeof(double));
        local_t = csecond() - local_t;
        ctrl->pin_alloc_t += local_t;
        local_t = csecond();
        Dtranspose(C_cpu, &(C[M_gpu * N]), M_cpu, N);
        local_t = csecond() - local_t;
        ctrl->transpose_t += local_t;
        ldC = M_cpu;
      }
      local_t = csecond();
      C_T = (double *)pin_malloc(M_gpu * N * sizeof(double));
      C_buffer = C_T;
      local_t = csecond() - local_t;
      ctrl->pin_alloc_t += local_t;
      local_t = csecond();
      Dtranspose(C_T, C, M_gpu, N);
      local_t = csecond() - local_t;
      ctrl->transpose_t += local_t;
      local_t = csecond();
      C_gpu = Dvec_transfer_gpu(C_T, M_gpu * N);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
    } else if (ctrl->C_mem == COL_MAJOR) {
      if (pred->cblas_target == CblasRowMajor)
        error(
            "hybrid_dgemm_Msplit -> pred->cblas_target == CblasRowMajor and "
            "pred->C_mem == COL_MAJOR -> Mode not allowed.");
      else if (pred->cblas_target == CblasColMajor) {
        C_cpu = &(C[M_gpu]);
        ldC = M;
      }
      local_t = csecond();
      C_gpu = Dvec_chunk_transfer_gpu(C, N, M_gpu, M);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
    }

  } else if (pred->Cadd_device_id == 0) {
    cpu_beta = beta;
    local_t = csecond();
    C_gpu = (double *)gpu_malloc(M_gpu * N * sizeof(double));
    C_buffer = (double *)pin_malloc(M_gpu * N * sizeof(double));
    local_t = csecond() - local_t;
    ctrl->pin_alloc_t += local_t;

    switch (ctrl->C_mem) {
      case (ROW_MAJOR):
        if (pred->cblas_target == CblasRowMajor) {
          C_cpu = &(C[M_gpu * N]);
          ldC = N;
        } else if (pred->cblas_target == CblasColMajor) {
          local_t = csecond();
          C_cpu = (double *)pin_malloc(M_cpu * N * sizeof(double));
          local_t = csecond() - local_t;
          ctrl->pin_alloc_t += local_t;
          local_t = csecond();
          Dtranspose(C_cpu, &(C[M_gpu * N]), M_cpu, N);
          local_t = csecond() - local_t;
          ctrl->transpose_t += local_t;
          ldC = M_cpu;
        }
        break;
      case (COL_MAJOR):
        if (pred->cblas_target == CblasRowMajor)
          error(
              "hybrid_dgemm_Msplit -> pred->cblas_target == CblasRowMajor and "
              "pred->C_mem == COL_MAJOR -> Mode not allowed");
        else if (pred->cblas_target == CblasColMajor) {
          error(
              "hybrid_dgemm_Msplit -> pred->cblas_target == CblasColMajor and "
              "pred->C_mem == COL_MAJOR and pred->Cadd_device_id == 0 -> Mode "
              "not allowed");
        }
        break;
      default:
        error("hybrid_dgemm_Msplit -> A Unknown mem type");
    }

  } else if (pred->Cadd_device_id == 1) {
    error("hybrid_dgemm_Msplit -> pred->Cadd_device_id == 1 Unimplemented.");
  }

  if (!ldA || !ldB || !ldC || !d_ldA || !d_ldB || !d_ldC)
    error("hybrid_dgemm_Msplit -> Some ld_dim were not defined correctly (=0)");

  gpu_timer_start(cuda_timer);

  stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M_gpu, N, K, &alpha, A_gpu,
                     d_ldA, B_gpu, d_ldB, &gpu_beta, C_gpu, d_ldC);
  gpu_timer_stop(cuda_timer);

  if (pred->Cadd_device_id == 0) {
    local_t = csecond();
    cblas_dscal(N * M_gpu, beta, C, 1);
    local_t = csecond() - local_t;
    ctrl->reduce_t = local_t;
  }

  local_t = csecond();
  cblas_dgemm(pred->cblas_target, cpu_op_A, cpu_op_B, M_cpu, N, K, alpha, A_cpu,
              ldA, B_cpu, ldB, cpu_beta, C_cpu, ldC);
  local_t = csecond() - local_t;
  ctrl->cpu_ex_t = local_t;
  cudaCheckErrors();

  ctrl->gpu_ex_t = (double)gpu_timer_get(cuda_timer) / 1000;

  if (ctrl->C_mem == ROW_MAJOR) {
    local_t = csecond();
    hipMemcpy(C_buffer, C_gpu, M_gpu * N * sizeof(double),
               hipMemcpyDeviceToHost);
    local_t = csecond() - local_t;
    ctrl->gather_t = local_t;
    if (pred->cblas_target == CblasRowMajor) {
      local_t = csecond();
      if (pred->Cadd_device_id == 0)
        Dtranspose_add(C, C_buffer, N, M_gpu);
      else
        Dtranspose(C, C_buffer, N, M_gpu);
      local_t = csecond() - local_t;
      ctrl->transpose_t += local_t;
    } else if (pred->cblas_target == CblasColMajor) {
      local_t = csecond();
      if (pred->Cadd_device_id == 0)
        Dtranspose_add(C, C_buffer, N, M_gpu);
      else
        Dtranspose(C, C_buffer, N, M_gpu);
      Dtranspose(&C[N * M_gpu], C_cpu, N, M_cpu);
      local_t = csecond() - local_t;
      ctrl->transpose_t += local_t;
    }
  } else if (ctrl->C_mem == COL_MAJOR) {
    if (pred->cblas_target == CblasRowMajor) {
      error("Should be thrown earlier(?)");
    }
    if (pred->cblas_target == CblasColMajor) {
      local_t = csecond();
      Dvec_chunk_copy_from_gpu(C, C_gpu, N, M_gpu, M);
      local_t = csecond() - local_t;
      ctrl->gather_t = local_t;
    }
  }

  printf(
      "Device overhead(M=%d, N=%d, K=%d) pin_alloc = %lf ms, scatter = %lf "
      "ms, "
      "transpose = %lf ms, gather = %lf ms, reduce = %lf ms\n",
      M, N, K, 1000 * ctrl->pin_alloc_t, 1000 * ctrl->scatter_t,
      1000 * ctrl->transpose_t, 1000 * ctrl->gather_t, 1000 * ctrl->reduce_t);

  printf("Hybrid Sgemm(M_cpu=%d, N=%d, K=%d) CPU time = ", M_cpu, N, K);
  report_results(ctrl->cpu_ex_t, (long)M_cpu * K * (2 * N + 1),
                 (long)(M_cpu * K + K * N + M_cpu * N * 2) *
                     sizeof(double));  //(M*N+(long)M*K*(3*N+1))
  printf("\n");

  printf("Hybrid Sgemm(M_gpu=%d, N=%d, K=%d) GPU time = ", M_gpu, N, K);
  report_results(ctrl->gpu_ex_t, (long)M_gpu * K * (2 * N + 1),
                 (long)(M_gpu * K + K * N + M_gpu * N * 2) *
                     sizeof(double));  //(M*N+(long)M*K*(3*N+1))
  printf("\n");

  // pin_free(C_buffer);

  // FIXME: implement correct frees with switches
  gpu_free(A_gpu);
  gpu_free(B_gpu);
  gpu_free(C_gpu);

  C_out = C;
  debug("<- hybrid_dgemm_Msplit()\n");
  return C_out;
}

double *hybrid_dgemm_Nsplit(control_p ctrl, predict_p pred, size_t M, size_t N,
                            size_t K, double alpha, double *A, double *B,
                            double beta, double *C) {
  debug("-> hybrid_dgemm_Nsplit()");
  if (ctrl->num_devices < 1)
    error(
        "hybrid_dgemm_Nsplit -> 0 or less devices? What are you trying to "
        "do...");
  else if (ctrl->num_devices > 2)
    error(
        "hybrid_dgemm_Nsplit -> Max 1 GPU + 1 CPU implemented (nice "
        "try,though).");

  if (pred->asynch_trans)
    error(
        "hybrid_dgemm_Nsplit -> asynch transactions not implemented yet "
        "(nice try,though).");

  if (!pred->N_split || pred->N_split >= N)
    error("hybrid_dgemm_Nsplit -> Full CPU/GPU versions do not belong here");
  if (!A) error("hybrid_dgemm_Nsplit -> A is not malloc'ed correctly");
  if (!B) error("hybrid_dgemm_Nsplit -> B is not malloc'ed correctly");
  if (beta != 0 && !C)
    error("hybrid_dgemm_Nsplit -> C is not malloc'ed correctly");
  debug(
      "hybrid_dgemm_Nsplit -> Trying your Matrix bounds (incomming "
      "segfaults)...");
  double test = A[M * K - 1];
  test = B[K * N - 1];
  if (beta != 0) test = C[M * N - 1];
  debug("hybrid_dgemm_Nsplit -> Passed.");
  double *C_out, local_t;

  double *A_cpu, *A_gpu, *B_cpu, *B_gpu, *C_cpu, *C_gpu;

  double *A_T, *B_T, *C_T, *C_buffer, cpu_beta = 0, gpu_beta = 0;

  size_t N_gpu = pred->N_split, N_cpu = N - pred->N_split, ldA = 0, ldB = 0,
         ldC = 0, d_ldA = 0, d_ldB = 0, d_ldC = 0;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans

  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);

  gpu_timer_p cuda_timer = gpu_timer_init();

  /// Setup A parts on host and device
  A_cpu = A;
  local_t = csecond();
  A_gpu = Dvec_transfer_gpu(A, M * K);
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  switch (ctrl->A_mem) {
    case (ROW_MAJOR):
      gpu_op_A = HIPBLAS_OP_T;
      d_ldA = K;
      ldA = K;
      if (pred->cblas_target == CblasRowMajor)
        cpu_op_A = CblasNoTrans;
      else if (pred->cblas_target == CblasColMajor)
        cpu_op_A = CblasTrans;
      break;
    case (COL_MAJOR):
      gpu_op_A = HIPBLAS_OP_N;
      d_ldA = M;
      ldA = M;
      if (pred->cblas_target == CblasRowMajor)
        cpu_op_A = CblasTrans;
      else if (pred->cblas_target == CblasColMajor)
        cpu_op_A = CblasNoTrans;
      break;
    default:
      error("hybrid_dgemm_Nsplit -> Unreachable default reached ");
  }

  /// Setup B parts on host and device

  switch (ctrl->B_mem) {
    case (COL_MAJOR):
      local_t = csecond();
      B_gpu = Dvec_transfer_gpu(B, N_gpu * K);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
      gpu_op_B = HIPBLAS_OP_N;
      d_ldB = K;
      B_cpu = &(B[N_gpu * K]);
      ldB = K;
      if (pred->cblas_target == CblasRowMajor)
        cpu_op_B = CblasTrans;
      else if (pred->cblas_target == CblasColMajor)
        cpu_op_B = CblasNoTrans;
      break;
    case (ROW_MAJOR):
      gpu_op_B = HIPBLAS_OP_T;
      d_ldB = N_gpu;

      if (pred->cblas_target == CblasRowMajor) {
        B_cpu = &(A[N_gpu]);
        ldB = N;
        cpu_op_B = CblasNoTrans;
      }

      else if (pred->cblas_target == CblasColMajor)
        error(
            "hybrid_dgemm_Nsplit -> pred->cblas_target == CblasColMajor and "
            "pred->B_mem == ROW_MAJOR -> Mode not allowed.");

      local_t = csecond();
      B_gpu = Dvec_chunk_transfer_gpu(B, K, N_gpu, N);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;

      break;
    default:
      error("hybrid_dgemm_Nsplit -> Unreachable default reached ");
  }

  /// Setup C parts on host and device
  d_ldC = M;

  if (!beta) {
    local_t = csecond();
    C_gpu = (double *)gpu_malloc(M * N_gpu * sizeof(double));
    C_buffer = (double *)pin_malloc(M * N_gpu * sizeof(double));
    if (!C)
      // TODO: Can we use only as much space as required?
      error("Unallocated input C not implemented");
    local_t = csecond() - local_t;
    ctrl->pin_alloc_t += local_t;

    if (ctrl->C_mem == COL_MAJOR) {
      if (pred->cblas_target == CblasColMajor) {
        C_cpu = &(C[N_gpu * M]);
        ldC = M;
      } else if (pred->cblas_target == CblasRowMajor)
        error(
            "hybrid_dgemm_Msplit -> pred->cblas_target == CblasRowMajor and "
            "pred->C_mem == COL_MAJOR -> Mode not allowed.");

    } else if (ctrl->C_mem == ROW_MAJOR) {
      if (pred->cblas_target == CblasRowMajor) {
        C_cpu = &(C[N_gpu]);
        ldC = N;
      } else if (pred->cblas_target == CblasColMajor) {
        local_t = csecond();
        C_cpu = (double *)pin_malloc(M * N_cpu * sizeof(double));
        local_t = csecond() - local_t;
        ctrl->pin_alloc_t += local_t;
        ldC = M;
      }
    }
  }
  /*else if (pred->Cadd_device_id == -1) {
      cpu_beta = gpu_beta = beta;
      if (ctrl->C_mem == ROW_MAJOR) {
        C_cpu = &(C[M_gpu * N]);
        if (pred->cblas_target == CblasRowMajor)
          ldC = N;
        else if (pred->cblas_target == CblasColMajor) {
          local_t = csecond();
          C_cpu = (double *)pin_malloc(M_cpu * N * sizeof(double));
          local_t = csecond() - local_t;
          ctrl->pin_alloc_t += local_t;
          local_t = csecond();
          Dtranspose(C_cpu, &(C[M_gpu * N]), M_cpu, N);
          local_t = csecond() - local_t;
          ctrl->transpose_t += local_t;
          ldC = M_cpu;
        }
        local_t = csecond();
        C_T = (double *)pin_malloc(M_gpu * N * sizeof(double));
        C_buffer = C_T;
        local_t = csecond() - local_t;
        ctrl->pin_alloc_t += local_t;
        local_t = csecond();
        Dtranspose(C_T, C, M_gpu, N);
        local_t = csecond() - local_t;
        ctrl->transpose_t += local_t;
        local_t = csecond();
        C_gpu = Dvec_transfer_gpu(C_T, M_gpu * N);
        local_t = csecond() - local_t;
        ctrl->scatter_t += local_t;
      } else if (ctrl->C_mem == COL_MAJOR) {
        if (pred->cblas_target == CblasRowMajor) error("hybrid_dgemm_Msplit ->
  pred->cblas_target == CblasRowMajor and " "pred->C_mem == COL_MAJOR -> Mode
  not allowed."); else if (pred->cblas_target == CblasColMajor) { C_cpu =
  &(C[M_gpu]); ldC = M;
        }
        local_t = csecond();
        C_gpu = Dvec_chunk_transfer_gpu(C, N, M_gpu, M);
        local_t = csecond() - local_t;
        ctrl->scatter_t += local_t;
      }

    } else if (pred->Cadd_device_id == 0) {

          cpu_beta = beta;
      local_t = csecond();
      C_gpu = (double *)gpu_malloc(M_gpu * N * sizeof(double));
      C_buffer = (double *)pin_malloc(M_gpu * N * sizeof(double));
      local_t = csecond() - local_t;
              ctrl->pin_alloc_t += local_t;

          switch (ctrl->C_mem) {
            case (ROW_MAJOR):
        if (pred->cblas_target == CblasRowMajor) {
          C_cpu = &(C[M_gpu * N]);
          ldC = N;
        } else if (pred->cblas_target == CblasColMajor) {
          local_t = csecond();
          C_cpu = (double *)pin_malloc(M_cpu * N * sizeof(double));
          local_t = csecond() - local_t;
          ctrl->pin_alloc_t += local_t;
          local_t = csecond();
          Dtranspose(C_cpu, &(C[M_gpu * N]), M_cpu, N);
          local_t = csecond() - local_t;
          ctrl->transpose_t += local_t;
          ldC = M_cpu;
        }
              break;
            case (COL_MAJOR):
        if (pred->cblas_target == CblasRowMajor) error("hybrid_dgemm_Msplit ->
  pred->cblas_target == CblasRowMajor and " "pred->C_mem == COL_MAJOR -> Mode
  not allowed"); else if (pred->cblas_target == CblasColMajor) {
  error("hybrid_dgemm_Msplit -> pred->cblas_target == CblasColMajor and "
            "pred->C_mem == COL_MAJOR and pred->Cadd_device_id == 0 -> Mode not
  allowed");
        }
              break;
            default:
              error("hybrid_dgemm_Msplit -> A Unknown mem type");
          }

    } else if (pred->Cadd_device_id == 1) {
      error(
          "hybrid_dgemm_Msplit -> pred->Cadd_device_id == 1 Unimplemented.");
    }
  */

  if (!ldA || !ldB || !ldC || !d_ldA || !d_ldB || !d_ldC)
    error("hybrid_dgemm_Msplit -> Some ld_dim were not defined correctly (=0)");

  gpu_timer_start(cuda_timer);

  stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M, N_gpu, K, &alpha, A_gpu,
                     d_ldA, B_gpu, d_ldB, &gpu_beta, C_gpu, d_ldC);
  gpu_timer_stop(cuda_timer);

  if (pred->Cadd_device_id == 0) {
    local_t = csecond();
    cblas_dscal(N_gpu * M, beta, C, 1);
    local_t = csecond() - local_t;
    ctrl->reduce_t = local_t;
  }

  local_t = csecond();
  cblas_dgemm(pred->cblas_target, cpu_op_A, cpu_op_B, M, N_cpu, K, alpha, A_cpu,
              ldA, B_cpu, ldB, cpu_beta, C_cpu, ldC);
  local_t = csecond() - local_t;
  ctrl->cpu_ex_t = local_t;
  cudaCheckErrors();

  ctrl->gpu_ex_t = (double)gpu_timer_get(cuda_timer) / 1000;

  if (ctrl->C_mem == ROW_MAJOR) {
    local_t = csecond();
    hipMemcpy(C_buffer, C_gpu, M * N_gpu * sizeof(double),
               hipMemcpyDeviceToHost);
    local_t = csecond() - local_t;
    ctrl->gather_t = local_t;
    if (pred->cblas_target == CblasRowMajor) {
      local_t = csecond();
      if (pred->Cadd_device_id == 0)
        Dtranspose_add(C, C_buffer, N, M_gpu);
      else
        Dtranspose(C, C_buffer, N, M_gpu);
      local_t = csecond() - local_t;
      ctrl->transpose_t += local_t;
    } else if (pred->cblas_target == CblasColMajor) {
      local_t = csecond();
      if (pred->Cadd_device_id == 0)
        Dtranspose_add(C, C_buffer, N, M_gpu);
      else
        Dtranspose(C, C_buffer, N, M_gpu);
      Dtranspose(&C[N * M_gpu], C_cpu, N, M_cpu);
      local_t = csecond() - local_t;
      ctrl->transpose_t += local_t;
    }
  } else if (ctrl->C_mem == COL_MAJOR) {
    if (pred->cblas_target == CblasRowMajor) {
      error("Should be thrown earlier(?)");
    }
    if (pred->cblas_target == CblasColMajor) {
      local_t = csecond();
      Dvec_chunk_copy_from_gpu(C, C_gpu, N, M_gpu, M);
      local_t = csecond() - local_t;
      ctrl->gather_t = local_t;
    }
  }

  printf(
      "Device overhead(M=%d, N=%d, K=%d) pin_alloc = %lf ms, scatter = %lf "
      "ms, "
      "transpose = %lf ms, gather = %lf ms, reduce = %lf ms\n",
      M, N, K, 1000 * ctrl->pin_alloc_t, 1000 * ctrl->scatter_t,
      1000 * ctrl->transpose_t, 1000 * ctrl->gather_t, 1000 * ctrl->reduce_t);

  printf("Hybrid Sgemm(M_cpu=%d, N=%d, K=%d) CPU time = ", M_cpu, N, K);
  report_results(ctrl->cpu_ex_t, (long)M_cpu * K * (2 * N + 1),
                 (long)(M_cpu * K + K * N + M_cpu * N * 2) *
                     sizeof(double));  //(M*N+(long)M*K*(3*N+1))
  printf("\n");

  printf("Hybrid Sgemm(M_gpu=%d, N=%d, K=%d) GPU time = ", M_gpu, N, K);
  report_results(ctrl->gpu_ex_t, (long)M_gpu * K * (2 * N + 1),
                 (long)(M_gpu * K + K * N + M_gpu * N * 2) *
                     sizeof(double));  //(M*N+(long)M*K*(3*N+1))
  printf("\n");

  // pin_free(C_buffer);

  // FIXME: implement correct frees with switches
  gpu_free(A_gpu);
  gpu_free(B_gpu);
  gpu_free(C_gpu);

  C_out = C;
  debug("<- hybrid_dgemm_Msplit()\n");
  return C_out;
}

int main(const int argc, const char *argv[]) {
  // print_devices();

  /*
    double *test, *test_T;

    test = Dvec_init_pinned(25, 42);
    test_T = Dvec_init_pinned(25, 0);

    for (int i = 0; i <5; i++){
            for (int j = 0; j <5; j++) printf("%0.3lf ", test[5*i +j]);
            printf("\n");
    }
    printf("\n");
    printf("\n");

    int s1 = 2, s2 = 5;
    Dtranspose(test_T, test, s1, s2);

    for (int i = 0; i <s1; i++){
            for (int j = 0; j <s2; j++) printf("%0.3lf ", test[s2*i +j]);
            printf("\n");
    }
    printf("\n");
    for (int i = 0; i <s1*s2; i++)printf("%0.3lf ", test_T[i]);
    printf("\n");


    exit(1);
  */

  // Arguments: ( M N K A_mem{0,1} B_mem{0,1} C_mem{0,1} alpha) (beta
  // add_device) M_split N_split K_split cblas_mem{0,1}

  double alpha, beta;

  size_t M, N, K;

  predict_p main_pred = (predict_p)malloc(sizeof(struct pred_str));
  control_p main_ctrl = (control_p)malloc(sizeof(struct control_str));

  main_ctrl->num_devices = 2;

  M = 100;
  K = 200;
  N = 300;
  main_ctrl->A_mem = ROW_MAJOR;
  main_ctrl->B_mem = ROW_MAJOR;
  main_ctrl->C_mem = ROW_MAJOR;
  alpha = 1.1;
  beta = 0;
  main_pred->Cadd_device_id = -1;
  main_pred->asynch_trans = 0;

  int ctr = 1;
  switch (argc) {
    case (14):
      M = atoi(argv[ctr++]);
      N = atoi(argv[ctr++]);
      K = atoi(argv[ctr++]);
      if (atoi(argv[ctr++]))
        main_ctrl->A_mem = COL_MAJOR;
      else
        main_ctrl->A_mem = ROW_MAJOR;
      if (atoi(argv[ctr++]))
        main_ctrl->B_mem = COL_MAJOR;
      else
        main_ctrl->B_mem = ROW_MAJOR;
      if (atoi(argv[ctr++]))
        main_ctrl->C_mem = COL_MAJOR;
      else
        main_ctrl->C_mem = ROW_MAJOR;
      alpha = atof(argv[ctr++]);
    case (7):
      beta = atof(argv[ctr++]);
      main_pred->Cadd_device_id = atoi(argv[ctr++]);
    case (5):
      main_pred->M_split = atoi(argv[ctr++]);
      main_pred->N_split = atoi(argv[ctr++]);
      main_pred->K_split = atoi(argv[ctr++]);
      if (atoi(argv[ctr++]))
        main_pred->cblas_target = CblasColMajor;
      else
        main_pred->cblas_target = CblasRowMajor;
      break;
    default:
      error("Incorrect input arguments");
  }

  printf("\nMatrix details: A(%s) B(%s) C(%s) -> M = %d, N = %d, K = %d\n",
         print_mem(main_ctrl->A_mem), print_mem(main_ctrl->B_mem),
         print_mem(main_ctrl->C_mem), M, N, K);
  printf("Constants: alpha = %lf, beta = %lf\n", alpha, beta);
  printf(
      "Predicted values: M_split = %d, N_split = %d, K_split = %d Cadd_device "
      "= %d, cblas_target = %d\n\n",
      main_pred->M_split, main_pred->N_split, main_pred->K_split,
      main_pred->Cadd_device_id, atoi(argv[ctr - 1]));
  double transpose_timer, cpu_timer = csecond();
  float gpu_preproc_t = 0, gpu_comp_t = 0, gpu_reduce_t = 0;
  gpu_timer_p cuda_timer = gpu_timer_init();

  // hipStream_t stream1, stream2;
  // hipStreamCreate(&stream1);
  // hipStreamCreate (&stream2);

  double *A, *B, *C, *C_comp, *C_buffer, *d_A, *d_B, *d_C, *C_T;
  size_t ldA, ldB, ldC;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans
  CBLAS_LAYOUT cblas_layout;

  if (main_ctrl->A_mem == COL_MAJOR)
    ldA = M;
  else
    ldA = K;
  if (main_ctrl->B_mem == COL_MAJOR)
    ldB = K;
  else
    ldB = N;

  if (main_ctrl->C_mem == COL_MAJOR) {
    cblas_layout = CblasColMajor;
    ldC = M;
    if (main_ctrl->A_mem == COL_MAJOR) {
      cpu_op_A = CblasNoTrans;
      gpu_op_A = HIPBLAS_OP_N;
    } else {
      cpu_op_A = CblasTrans;
      gpu_op_A = HIPBLAS_OP_T;
    }
    if (main_ctrl->B_mem == COL_MAJOR) {
      cpu_op_B = CblasNoTrans;
      gpu_op_B = HIPBLAS_OP_N;
    } else {
      cpu_op_B = CblasTrans;
      gpu_op_B = HIPBLAS_OP_T;
    }
  } else {
    cblas_layout = CblasRowMajor;
    ldC = N;
    if (main_ctrl->A_mem == ROW_MAJOR) {
      cpu_op_A = CblasNoTrans;
      gpu_op_A = HIPBLAS_OP_T;
    } else {
      cpu_op_A = CblasTrans;
      gpu_op_A = HIPBLAS_OP_N;
    }
    if (main_ctrl->B_mem == ROW_MAJOR) {
      cpu_op_B = CblasNoTrans;
      gpu_op_B = HIPBLAS_OP_T;
    } else {
      cpu_op_B = CblasTrans;
      gpu_op_B = HIPBLAS_OP_N;
    }
  }

  A = Dvec_init_pinned(M * K, 42);
  B = Dvec_init_pinned(K * N, 42);
  C = Dvec_init_pinned(M * N, 42);
  C_buffer = Dvec_init_host(M * N, 0);
  C_comp = Dvec_init_host(M * N, 0);

  Dvec_copy(C_comp, C, M * N);
  Dvec_copy(C_buffer, C, M * N);

  cpu_timer = csecond() - cpu_timer;
  printf("Initializing Arrays on host (M=%d, N=%d, K=%d) t_init = %lf ms\n", M,
         N, K, cpu_timer * 1000);
  cpu_timer = csecond();
  for (int i = 0; i < 10; i++)
    cblas_dgemm(cblas_layout, cpu_op_A, cpu_op_B, M, N, K, alpha, A, ldA, B,
                ldB, beta, C_comp, ldC);
  cpu_timer = csecond() - cpu_timer;
  printf("MKL Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
  report_results(cpu_timer / 10, (long)M * K * (2 * N + 1),
                 (long)(M * K + K * N + M * N * 2) * sizeof(double));
  printf("\n");

  transpose_timer = csecond();
  if (main_ctrl->C_mem == ROW_MAJOR) {
    C_T = (double *)pin_malloc(M * N * sizeof(double));
    Dtranspose(C_T, C, M, N);
  } else
    C_T = C;
  transpose_timer = csecond() - transpose_timer;

  gpu_timer_start(cuda_timer);
  d_A = Dvec_transfer_gpu(A, M * K);
  d_B = Dvec_transfer_gpu(B, K * N);
  d_C = Dvec_transfer_gpu(C_T, M * N);
  gpu_timer_stop(cuda_timer);
  gpu_preproc_t = gpu_timer_get(cuda_timer);

  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  // hipblasSetStream(handle, stream1);
  gpu_timer_start(cuda_timer);

  for (int i = 0; i < 10; i++) {
    stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M, N, K, &alpha, d_A, ldA,
                       d_B, ldB, &beta, d_C, M);
    hipDeviceSynchronize();
  }

  gpu_timer_stop(cuda_timer);
  cudaCheckErrors();
  gpu_comp_t = gpu_timer_get(cuda_timer);
  gpu_timer_start(cuda_timer);
  hipMemcpy(C_T, d_C, M * N * sizeof(double), hipMemcpyDeviceToHost);
  gpu_timer_stop(cuda_timer);
  transpose_timer = csecond() - transpose_timer;
  if (main_ctrl->C_mem == ROW_MAJOR) {
    Dtranspose(C, C_T, N, M);
    pin_free(C_T);
  } else
    C = C_T;
  transpose_timer = csecond() - transpose_timer;
  gpu_reduce_t = gpu_timer_get(cuda_timer);

  printf(
      "\nDevice overhead(M=%d, N=%d, K=%d) transpose = %lf ms, t_preproc = %lf "
      "ms, t_reduce = %lf ms\n",
      M, N, K, 1000 * transpose_timer, gpu_preproc_t, gpu_reduce_t);

  printf("CUDA Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
  report_results((double)gpu_comp_t / 1000.0 / 10, (long)M * K * (2 * N + 1),
                 (long)(M * K + K * N + M * N * 2) *
                     sizeof(double));  //(M*N+(long)M*K*(3*N+1))
  printf("\n");

  Dtest_equality(C_comp, C, M * N);
  printf("\n");

  gpu_free(d_A);
  gpu_free(d_B);
  gpu_free(d_C);
  pin_free(C);

  C = Dvec_init_pinned(M * N, 0);
  C_T = Dvec_init_pinned(M * N, 0);
  Dvec_copy(C, C_buffer, M * N);
  Dtranspose(C_T, C, M, N);
  Dvec_copy(C_comp, C_buffer, M * N);
  cblas_dgemm(cblas_layout, cpu_op_A, cpu_op_B, M, N, K, alpha, A, ldA, B, ldB,
              beta, C_comp, ldC);

  if (!main_pred->M_split + !main_pred->N_split + !main_pred->K_split < 2)
    error("split more than one dim for 2 devices.");

  if (main_pred->M_split) {
    if (main_pred->M_split == M) {
      C = gpu_dgemm(main_ctrl, main_pred, M, N, K, alpha, A, B, beta, C);

      printf("Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
      report_results(fmax(main_ctrl->cpu_ex_t, main_ctrl->gpu_ex_t) +
                         main_ctrl->scatter_t + main_ctrl->pin_alloc_t +
                         main_ctrl->transpose_t + main_ctrl->gather_t +
                         main_ctrl->reduce_t,
                     (long)M * K * (2 * N + 1),
                     (long)(M * K + K * N + M * N * 2) *
                         sizeof(double));  //(M*N+(long)M*K*(3*N+1))
      printf("\n");

      Dtest_equality(C_comp, C, M * N);

    } else {
      C = hybrid_dgemm_Msplit(main_ctrl, main_pred, M, N, K, alpha, A, B, beta,
                              C);

      printf("Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
      report_results(fmax(main_ctrl->cpu_ex_t, main_ctrl->gpu_ex_t) +
                         main_ctrl->scatter_t + main_ctrl->pin_alloc_t +
                         main_ctrl->transpose_t + main_ctrl->gather_t +
                         main_ctrl->reduce_t,
                     (long)M * K * (2 * N + 1),
                     (long)(M * K + K * N + M * N * 2) *
                         sizeof(double));  //(M*N+(long)M*K*(3*N+1))
      printf("\n");

      Dtest_equality(C_comp, C, M * N);
    }
  } else if (main_pred->N_split) {
    if (main_pred->N_split == N) {
      C = gpu_dgemm(main_ctrl, main_pred, M, N, K, alpha, A, B, beta, C);

      printf("Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
      report_results(fmax(main_ctrl->cpu_ex_t, main_ctrl->gpu_ex_t) +
                         main_ctrl->scatter_t + main_ctrl->pin_alloc_t +
                         main_ctrl->transpose_t + main_ctrl->gather_t +
                         main_ctrl->reduce_t,
                     (long)M * K * (2 * N + 1),
                     (long)(M * K + K * N + M * N * 2) *
                         sizeof(double));  //(M*N+(long)M*K*(3*N+1))
      printf("\n");

      Dtest_equality(C_comp, C, M * N);

    } else {
      C = hybrid_dgemm_Nsplit(main_ctrl, main_pred, M, N, K, alpha, A, B, beta,
                              C);

      printf("Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
      report_results(fmax(main_ctrl->cpu_ex_t, main_ctrl->gpu_ex_t) +
                         main_ctrl->scatter_t + main_ctrl->pin_alloc_t +
                         main_ctrl->transpose_t + main_ctrl->gather_t +
                         main_ctrl->reduce_t,
                     (long)M * K * (2 * N + 1),
                     (long)(M * K + K * N + M * N * 2) *
                         sizeof(double));  //(M*N+(long)M*K*(3*N+1))
      printf("\n");

      Dtest_equality(C_comp, C, M * N);
    }
  }

  /*
          debug("executing hybrid GPU-CPU (N)");
          size_t N_gpu = N_split, N_cpu = N - N_split;
          if (ghost_beta) {
            debug("executing with ghost_beta !=0");

            transpose_timer = csecond();
            double *B_T = (double *)pin_malloc(K * N * sizeof(double));
            Dtranspose(B_T, B, K, N);
            double *C_T = (double *)pin_malloc(M * N * sizeof(double));
            Dtranspose(C_T, C, M, N);
            transpose_timer = csecond() - transpose_timer;

            gpu_timer_start(cuda_timer);
            d_A = Dvec_transfer_gpu(A, M * K);
            d_B = Dvec_transfer_gpu(B_T, K * N_gpu);
            d_C = Dvec_transfer_gpu(C_T, M * N_gpu);
            gpu_timer_stop(cuda_timer);
            gpu_preproc_t = gpu_timer_get(cuda_timer);

            gpu_timer_start(cuda_timer);

            stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N_gpu, K,
                               &alpha, d_A, K, d_B, K, &beta, d_C, M);
            cpu_timer = csecond();
            cblas_dgemm(CblasColMajor, CblasTrans, CblasNoTrans, M, N_cpu, K,
      alpha, A, K, &(B_T[N_gpu * K]), K, beta, &(C_T[M * N_gpu]), M); cpu_timer
  = csecond() - cpu_timer; cudaCheckErrors();

            gpu_timer_stop(cuda_timer);
            gpu_comp_t = gpu_timer_get(cuda_timer);
            gpu_timer_start(cuda_timer);
            hipMemcpy(C_T, d_C, M * N_gpu * sizeof(double),
                       hipMemcpyDeviceToHost);
            gpu_timer_stop(cuda_timer);
            gpu_reduce_t = gpu_timer_get(cuda_timer);

            transpose_timer = csecond() - transpose_timer;
            Dtranspose(C, C_T, N, M);
            transpose_timer = csecond() - transpose_timer;

            printf(
                "Device overhead(M=%d, N=%d, K=%d) transpose = %lf ms, t_preproc
  "
                "= "
                "%lf ms, t_reduce = %lf ms\n",
                M, N, K, 1000 * transpose_timer, gpu_preproc_t, gpu_reduce_t);

            printf("Hybrid Sgemm(M=%d, N_cpu=%d, K=%d) CPU time = ", M, N_cpu,
  K); report_results((double)cpu_timer, (long)M * K * (2 * N_cpu + 1), (long)(M
  * K + K * N_cpu + M * N_cpu * 2) * sizeof(double));  //(M*N+(long)M*K*(3*N+1))
            printf("\n");

            printf("Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
            report_results((double)gpu_comp_t / 1000.0 + transpose_timer +
                               (double)gpu_preproc_t / 1000 +
                               (double)gpu_reduce_t / 1000,
                           (long)M * K * (2 * N + 1),
                           (long)(M * K + K * N + M * N * 2) *
                               sizeof(double));  //(M*N+(long)M*K*(3*N+1))
            printf("\n");

            Dtest_equality(C_comp, C, M * N);

            gpu_free(d_A);
            gpu_free(d_B);
            gpu_free(d_C);
            pin_free(B_T);
            pin_free(C_T);
          } else {
            debug("executing with ghost_beta = 0");
            gpu_timer_start(cuda_timer);
            double dev_beta = 0.0;

            transpose_timer = csecond();
            double *B_T = (double *)pin_malloc(K * N * sizeof(double));
            Dtranspose(B_T, B, K, N);
            double *C_T = (double *)pin_malloc(M * N * sizeof(double));
            Dtranspose(C_T, C, M, N);
            transpose_timer = csecond() - transpose_timer;

            gpu_timer_start(cuda_timer);
            d_A = Dvec_transfer_gpu(A, M * K);
            d_B = Dvec_transfer_gpu(B_T, K * N_gpu);
            d_C = (double *)gpu_malloc(M * N_gpu * sizeof(double));
            gpu_timer_stop(cuda_timer);
            gpu_preproc_t = gpu_timer_get(cuda_timer);

            hipblasHandle_t handle;
            hipblasStatus_t stat = hipblasCreate(&handle);
            // hipblasSetStream(handle, stream1);
            gpu_timer_start(cuda_timer);

            stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N_gpu, K,
                               &alpha, d_A, K, d_B, K, &dev_beta, d_C, M);
            cblas_dscal(M * N, beta, C, 1);
            cpu_timer = csecond();
            cblas_dgemm(CblasColMajor, CblasTrans, CblasNoTrans, M, N_cpu, K,
      alpha, A, K, &(B_T[N_gpu * K]), K, 0.0, &(C_T[M * N_gpu]), M); cpu_timer =
      csecond() - cpu_timer; cudaCheckErrors();

            gpu_timer_stop(cuda_timer);
            gpu_comp_t = gpu_timer_get(cuda_timer);
            gpu_timer_start(cuda_timer);
            hipMemcpy(C_T, d_C, M * N_gpu * sizeof(double),
                       hipMemcpyDeviceToHost);
            gpu_timer_stop(cuda_timer);
            gpu_reduce_t = gpu_timer_get(cuda_timer);

            transpose_timer = csecond() - transpose_timer;
            Dtranspose_add(C, C_T, N, M);
            transpose_timer = csecond() - transpose_timer;

            printf(
                "Device overhead(M=%d, N=%d, K=%d) transpose = %lf ms, t_preproc
  "
                "= "
                "%lf ms, t_reduce = %lf ms\n",
                M, N, K, 1000 * transpose_timer, gpu_preproc_t, gpu_reduce_t);

            printf("Hybrid Sgemm(M=%d, N_cpu=%d, K=%d) CPU time = ", M, N_cpu,
  K); report_results((double)cpu_timer, (long)M * K * (2 * N_cpu + 1), (long)(M
  * K + K * N_cpu + M * N_cpu * 2) * sizeof(double));  //(M*N+(long)M*K*(3*N+1))
            printf("\n");

            printf("Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
            report_results((double)gpu_comp_t / 1000.0 + transpose_timer +
                               (double)gpu_preproc_t / 1000 +
                               (double)gpu_reduce_t / 1000,
                           (long)M * K * (2 * N + 1),
                           (long)(M * K + K * N + M * N * 2) *
                               sizeof(double));  //(M*N+(long)M*K*(3*N+1))
            printf("\n");

            Dtest_equality(C_comp, C, M * N);

            gpu_free(d_A);
            gpu_free(d_B);
            gpu_free(d_C);
            pin_free(B_T);
            pin_free(C_T);
          }
        }
      }

  else if (K_split) {
        if (K_split == K) {
          debug("executing solely on GPU but with Κ spliting (?)");
          debug("Not gonna copy the same code, a goto is preffered");
          K_split = 0;
          M_split = M;
          goto return_point;
        } else {
          debug("executing hybrid GPU-CPU (K)");
          size_t K_gpu = K_split, K_cpu = K - K_split;
          if (ghost_beta) {
            debug("executing with ghost_beta !=0");
            debug("...you would think but that would calculate C = aAB + 2bC");
            debug("our friend goto will save us");
            goto just_bellow;
          } else {
          just_bellow:
            debug("executing with ghost_beta = 0");
            gpu_timer_start(cuda_timer);
            double dev_beta = 0.0;

            transpose_timer = csecond();
            double *A_T = (double *)pin_malloc(M * K * sizeof(double));
            Dtranspose(A_T, A, M, K);
            double *reduce_C;
            transpose_timer = csecond() - transpose_timer;

            gpu_timer_start(cuda_timer);
            d_A = Dvec_transfer_gpu(A_T, M * K_gpu);
            d_B = Dvec_transfer_gpu(B, K_gpu * N);
            d_C = (double *)gpu_malloc(M * N * sizeof(double));
            gpu_timer_stop(cuda_timer);
            gpu_preproc_t = gpu_timer_get(cuda_timer);

            gpu_timer_start(cuda_timer);

            stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, M, N, K_gpu,
                               &alpha, d_A, M, d_B, N, &dev_beta, d_C, M);
            reduce_C = (double *)pin_malloc(M * N * sizeof(double));
            cpu_timer = csecond();
            cblas_dgemm(CblasRowMajor, CblasTrans, CblasNoTrans, M, N, K_cpu,
      alpha,
                        &(A_T[M * K_gpu]), M, &(B[N * K_gpu]), N, beta, C, N);
            cpu_timer = csecond() - cpu_timer;
            cudaCheckErrors();

            gpu_timer_stop(cuda_timer);
            gpu_comp_t = gpu_timer_get(cuda_timer);
            gpu_timer_start(cuda_timer);
            hipMemcpy(reduce_C, d_C, M * N * sizeof(double),
                       hipMemcpyDeviceToHost);
            gpu_timer_stop(cuda_timer);
            gpu_reduce_t = gpu_timer_get(cuda_timer);

            transpose_timer = csecond() - transpose_timer;
            Dtranspose_add(C, reduce_C, N, M);
            transpose_timer = csecond() - transpose_timer;

            printf(
                "Device overhead(M=%d, N=%d, K=%d) transpose = %lf ms, t_preproc
  "
                "= "
                "%lf ms, t_reduce = %lf ms\n",
                M, N, K, 1000 * transpose_timer, gpu_preproc_t, gpu_reduce_t);

            printf("Hybrid Sgemm(M=%d, N=%d, K_cpu=%d) CPU time = ", M, N,
  K_cpu); report_results((double)cpu_timer, (long)M * K_cpu * (2 * N + 1),
                           (long)(M * K_cpu + K_cpu * N + M * N * 2) *
                               sizeof(double));  //(M*N+(long)M*K_cpu*(3*N+1))
            printf("\n");

            printf("Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
            report_results((double)gpu_comp_t / 1000.0 + transpose_timer +
                               (double)gpu_preproc_t / 1000 +
                               (double)gpu_reduce_t / 1000,
                           (long)M * K * (2 * N + 1),
                           (long)(M * K + K * N + M * N * 2) *
                               sizeof(double));  //(M*N+(long)M*K*(3*N+1))
            printf("\n");

            Dtest_equality(C_comp, C, M * N);

            gpu_free(d_A);
            gpu_free(d_B);
            gpu_free(d_C);
            pin_free(A_T);
            pin_free(reduce_C);
          }
        }
      } else {
        debug("Not spliting at all, execute the whole on host");

        cpu_timer = csecond();
        cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M, N, K, alpha,
  A, K, B, N, beta, C, N); cpu_timer = csecond() - cpu_timer; printf("MKL
      Sgemm(M=%d, N=%d, K=%d) ", M, N, K); report_results(cpu_timer, (long)M * K
  * (2 * N + 1), (long)(M * K + K * N + M * N * 2) * sizeof(double));
        printf("\n");
        Dtest_equality(C_comp, C, M * N);
      }
    */
}
