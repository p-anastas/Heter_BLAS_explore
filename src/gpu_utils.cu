///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cstdio>
#include "cpu_utils.hpp"
#include "gpu_utils.hpp"

void print_devices() {
  debug(" -> print_devices()");
  hipDeviceProp_t properties;
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipGetDeviceProperties(&properties, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", properties.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           properties.memoryClockRate / 1024);
    printf("  Memory Bus Width (bits): %d\n", properties.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0 * properties.memoryClockRate * (properties.memoryBusWidth / 8) /
               1.0e6);
    if (properties.major >= 3)
      printf("  Unified Memory support: YES\n\n");
    else
      printf("  Unified Memory support: NO\n\n");
  }
  debug(" <- print_devices()");
}

void test_bandwidth(size_t bytes) {
  debug(" -> test_bandwidth(bytes)");
  double exc_timer = 0, *host_vector = (double *)malloc(bytes),
         *host_vector1 = (double *)malloc(bytes);
  double *pinvector;
  hipHostAlloc(&pinvector, bytes, hipHostMallocDefault);
  double *devector;
  hipMalloc(&devector, bytes);
  // double * univector; hipMallocManaged(&univector, bytes);
  printf("Running Transaction benchmarks...\n\n");
  printf("Timing memcpy(%.3lf Mb)...", bytes * 1.0 / 1024 / 1024);
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++) memcpy(host_vector, host_vector1, bytes);
  exc_timer = csecond() - exc_timer;
  printf("\n -- Host to Host\t: ");
  report_bandwidth(exc_timer, bytes);
  free(host_vector1);
  /*
  exc_timer = 0;
    for (int i = 0; i < NR_ITER; i++) {
      hipMemPrefetchAsync(univector, bytes, 0);
      hipDeviceSynchronize();
      exc_timer = csecond() - exc_timer;
      memcpy(host_vector, univector, bytes);
      exc_timer = csecond() - exc_timer;
    }
  printf("\n -- Unified to Host\t: ");
  report_bandwidth(exc_timer,bytes);
  exc_timer = 0;
    for (int i = 0; i < NR_ITER; i++) {
      hipMemPrefetchAsync(univector, bytes, 0);
      hipDeviceSynchronize();
      exc_timer = csecond() - exc_timer;
      memcpy(univector, host_vector, bytes);
      exc_timer = csecond() - exc_timer;
    }
  printf("\n -- Host to Unified\t: ");
  report_bandwidth(exc_timer,bytes);
  */
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++) memcpy(host_vector, pinvector, bytes);
  exc_timer = csecond() - exc_timer;
  printf("\n -- Pinned H. to Host\t: ");
  report_bandwidth(exc_timer, bytes);
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++) memcpy(pinvector, host_vector, bytes);
  exc_timer = csecond() - exc_timer;
  printf("\n -- Host to Pinned H.\t: ");
  report_bandwidth(exc_timer, bytes);
  printf("\n\n");
  printf("Timing hipMemcpy(%.3lf Mb)...", bytes * 1.0 / 1024 / 1024);
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++) {
    hipMemcpy(devector, devector, bytes, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
  }
  exc_timer = csecond() - exc_timer;
  printf("\n -- Device to Device\t: ");
  report_bandwidth(exc_timer, bytes);
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++)
    hipMemcpy(host_vector, devector, bytes, hipMemcpyDeviceToHost);
  exc_timer = csecond() - exc_timer;
  printf("\n -- Device to Host\t: ");
  report_bandwidth(exc_timer, bytes);
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++)
    hipMemcpy(devector, host_vector, bytes, hipMemcpyHostToDevice);
  exc_timer = csecond() - exc_timer;
  printf("\n -- Host to Device\t: ");
  report_bandwidth(exc_timer, bytes);
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++)
    hipMemcpy(devector, pinvector, bytes, hipMemcpyHostToDevice);
  exc_timer = csecond() - exc_timer;
  printf("\n -- Pinned H. to Device\t: ");
  report_bandwidth(exc_timer, bytes);
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
  for (int i = 0; i < NR_ITER; i++)
    hipMemcpy(pinvector, devector, bytes, hipMemcpyDeviceToHost);
  exc_timer = csecond() - exc_timer;
  printf("\n -- Device to Pinned H.\t: ");
  report_bandwidth(exc_timer, bytes);
  /*
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
    for (int i = 0; i < NR_ITER; i++)
      hipMemcpy(univector, pinvector, bytes,
                 hipMemcpyHostToDevice);
      exc_timer = csecond() - exc_timer;
  printf("\n -- Pinned H. to Uni.\t: ");
  report_bandwidth(exc_timer,bytes);
  exc_timer = 0;
  exc_timer = csecond() - exc_timer;
    for (int i = 0; i < NR_ITER; i++)
      hipMemcpy(pinvector, univector, bytes,
                 hipMemcpyDeviceToHost);
      exc_timer = csecond() - exc_timer;
  printf("\n -- Uni. to Pinned H.\t: ");
  report_bandwidth(exc_timer,bytes);
  */
  printf("\n\n");
  free(host_vector);
  hipHostFree(pinvector);
  gpu_free(devector);

  debug(" <- test_bandwidth(bytes)");
}

void cudaCheckErrors() {
  hipError_t errSync = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void *gpu_alloc(size_t count) {
  void *ret;
  massert(hipMalloc(&ret, count) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
  return ret;
}

void gpu_free(void *gpuptr) {
  massert(hipFree(gpuptr) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
}

void gpu_showMem(char *message) {
  size_t free, total;
  massert(hipMemGetInfo(&free, &total) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
  printf("showMem(%s): %u free host_vector of %u MB \n", message,
         free / (1024 * 1024), total / (1024 * 1024));
}

gpu_timer_p gpu_timer_init() {
  gpu_timer_p timer = (gpu_timer_p)malloc(sizeof(struct gpu_timer));
  hipEventCreate(&timer->start);
  hipEventCreate(&timer->stop);
  return timer;
}

void gpu_timer_start(gpu_timer_p timer) { hipEventRecord(timer->start); }

void gpu_timer_stop(gpu_timer_p timer) { hipEventRecord(timer->stop); }

float gpu_timer_get(gpu_timer_p timer) {
  hipEventSynchronize(timer->stop);
  hipEventElapsedTime(&timer->ms, timer->start, timer->stop);
  return timer->ms;
}

float *Svec_init_pinned(size_t size, float val) {
  float *vec;
  hipHostMalloc(&vec, size * sizeof(float));
  for (size_t i = 0; i < size; i++) vec[i] = val;  //(float) Drandom(-1,1);
  return vec;
}

float *Svec_transfer_gpu(float *host_vec, size_t size) {
  float *dev_vec;
  hipMalloc(&dev_vec, size * sizeof(float));
  hipMemcpy(dev_vec, host_vec, size * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors();
  return dev_vec;
}
