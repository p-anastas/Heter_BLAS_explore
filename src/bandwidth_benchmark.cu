///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief The start of Zawarudo
///

#include <hip/hip_runtime.h>
#include <mkl.h>
#include "hipblas.h"

#include "cpu_utils.hpp"
#include "gpu_utils.hpp"

int main(const int argc, const char *argv[]) {
  double alpha, *src, *dest;

  size_t N_bytes, from, to, itterations = 1000;

  switch (argc) {
    case (4):
      N_bytes = atoi(argv[1]);
      from = atoi(argv[2]);
      to = atoi(argv[3]);
      break;
    default:
      error("Incorrect input arguments");
  }

  gpu_timer_p cuda_timer = gpu_timer_init();
  double total_t = 0;

  int count = 666;
  hipGetDeviceCount(&count);

  if (-1 == from) {
    fprintf(stderr, "Copying %d bytes from host...", N_bytes);
    src = Dvec_init_pinned(N_bytes, 42);
  } else if (from >= count || from < 0)
    error("Invalid source device");
  else {
    fprintf(stderr, "Copying %d bytes from device(%d)...", N_bytes, from);
    hipSetDevice(from);
    hipMalloc(&src, N_bytes);
  }

  if (-1 == to) {
    fprintf(stderr, "to host\n", N_bytes);
    dest = Dvec_init_pinned(N_bytes, 0);
  } else if (to >= count || to < 0)
    error("Invalid destination device");
  else {
    fprintf(stderr, "to device(%d)\n", to);
    hipSetDevice(to);
    hipMalloc(&dest, N_bytes);
  }

  gpu_timer_start(cuda_timer);
  if (-2 == from + to)
    for (int it = 0; it < itterations; it++) memcpy(dest, src, N_bytes);
  if (-1 == from)
    for (int it = 0; it < itterations; it++)
      hipMemcpy(dest, src, N_bytes, hipMemcpyHostToDevice);
  if (-1 == to)
    for (int it = 0; it < itterations; it++)
      hipMemcpy(dest, src, N_bytes, hipMemcpyDeviceToHost);
  else
    for (int it = 0; it < itterations; it++)
      hipMemcpy(dest, src, N_bytes, hipMemcpyDeviceToDevice);
  gpu_timer_stop(cuda_timer);
  total_t = gpu_timer_get(cuda_timer) / 1000;
  fprintf(stderr,
          "bandwidth(%d) benchmarked sucsessfully t = %lf ms ( %.3lf Gb/s "
          "%.15lf s/byte)\n",
          N_bytes, total_t * 1000 / itterations,
          1e-9 / (total_t / N_bytes / itterations),
          total_t / N_bytes / itterations);
  fprintf(stdout, "%d,%d,%d,%.15lf\n", N_bytes, from, to,
          total_t / itterations);
  return 0;
}
