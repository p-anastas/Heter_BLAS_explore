#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief The start of Zawarudo
///

#include <hip/hip_runtime.h>
#include <mkl.h>
#include "hipblas.h"

#include "cpu_utils.hpp"
#include "gpu_utils.hpp"

/// Inputs: A, B, C, alpha, beta, M, N, K, store format (Colmajor OR Rowmajor)
/// for A, B, C
/// TODO: Add lda, ldb, ldc of initial call in order to also predict cases for
/// sub-matrix gemm ( if ld_dim > dim etc)

/// Predictor inputs: M_split (OR) N_split (OR) K_split, <<cblas Layout (for
/// CPU), >>, C_add device
/// TODO: Also add asynch trans

/// Output: A, B intact, C dgemm result
/// TODO: No mem leaks (CPU and GPU)

/// Extras: Precise debug and parameter error checking
/// TODO: More than two devices

enum mem_layout { ROW_MAJOR = 0, COL_MAJOR };

const char *print_mem(mem_layout mem) {
  if (mem == ROW_MAJOR)
    return "Row major";
  else if (mem == COL_MAJOR)
    return "Col major";
  else
    return "ERROR";
}

typedef struct control_str {
  // For now assume device 0 is always the host
  size_t num_devices;
  mem_layout A_mem;
  mem_layout B_mem;
  mem_layout C_mem;
  double alloc_t = 0, scatter_t = 0, transpose_t = 0, cpu_ex_t = 0,
         gpu_ex_t = 0, gather_t = 0, reduce_t = 0;

} * control_p;

typedef struct pred_str {
  size_t M_split;
  size_t N_split;
  size_t K_split;
  size_t Cadd_device_id;
  // TODO: Not implemented
  int asynch_trans = 0;

} * predict_p;

double *gpu_dgemm(control_p ctrl, predict_p pred, size_t M, size_t N, size_t K,
                  double alpha, double *A, double *B, double beta, double *C) {
  debug("-> gpu_dgemm()");
  if (ctrl->num_devices < 1)
    error(
        "gpu_dgemm -> 0 or less devices? What are you trying to "
        "do...");
  else if (ctrl->num_devices > 2)
    error(
        "gpu_dgemm -> Max 1 GPU + 1 CPU implemented (nice "
        "try,though).");

  if (pred->asynch_trans)
    error(
        "gpu_dgemm -> asynch transactions not implemented yet "
        "(nice try,though).");

  if (!A) error("gpu_dgemm -> A is not malloc'ed correctly");
  if (!B) error("gpu_dgemm -> B is not malloc'ed correctly");
  if (beta != 0 && !C) error("gpu_dgemm -> C is not malloc'ed correctly");
  debug(
      "gpu_dgemm -> Trying your Matrix bounds (incomming "
      "segfaults)...");
  double test = A[M * K - 1];
  test = B[K * N - 1];
  if (beta != 0) test = C[M * N - 1];
  debug("gpu_dgemm -> Passed.");
  double *C_out, local_t;

  double *A_gpu, *B_gpu, *C_gpu;

  double *C_T, cpu_beta = 0, gpu_beta = 0;

  size_t M_gpu = pred->M_split, ldA = 0, ldB = 0, ldC = 0, d_ldA = 0, d_ldB = 0,
         d_ldC = 0;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans

  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);

  gpu_timer_p cuda_timer = gpu_timer_init();

  if (ctrl->A_mem == COL_MAJOR)
    ldA = M;
  else
    ldA = K;
  if (ctrl->B_mem == COL_MAJOR)
    ldB = K;
  else
    ldB = N;

  if (ctrl->C_mem == COL_MAJOR) {
    ldC = M;
    if (ctrl->A_mem == COL_MAJOR)
      gpu_op_A = HIPBLAS_OP_N;
    else
      gpu_op_A = HIPBLAS_OP_T;
    if (ctrl->B_mem == COL_MAJOR)
      gpu_op_B = HIPBLAS_OP_N;
    else
      gpu_op_B = HIPBLAS_OP_T;
  } else {
    ldC = N;
    if (ctrl->A_mem == ROW_MAJOR)
      gpu_op_A = HIPBLAS_OP_T;
    else
      gpu_op_A = HIPBLAS_OP_N;

    if (ctrl->B_mem == ROW_MAJOR)
      gpu_op_B = HIPBLAS_OP_T;
    else
      gpu_op_B = HIPBLAS_OP_N;
  }

  local_t = csecond();
  A_gpu = Dvec_transfer_gpu(A, M * K);
  B_gpu = Dvec_transfer_gpu(B, K * N);
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  local_t = csecond();
  C_T = (double *)pin_malloc(M * N * sizeof(double));
  local_t = csecond() - local_t;
  ctrl->alloc_t += local_t;

  if (pred->Cadd_device_id == -1) {
    gpu_beta = beta;
    local_t = csecond();
    if (ctrl->C_mem == ROW_MAJOR) Dtranspose(C_T, C, M, N);

    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;

    local_t = csecond();
    C_gpu = Dvec_transfer_gpu(C_T, M * N);
    local_t = csecond() - local_t;
    ctrl->scatter_t += local_t;
  } else if (pred->Cadd_device_id == 0) {
    local_t = csecond();
    C_gpu = (double *)gpu_malloc(M * N * sizeof(double));
    local_t = csecond() - local_t;
    ctrl->alloc_t += local_t;
  }

  else if (pred->Cadd_device_id == 1) {
    debug(
        "gpu_dgemm -> pred->Cadd_device_id == 1 is obsolete..all computations "
        "on gpu anyway");

    gpu_beta = beta;
    local_t = csecond();
    if (ctrl->C_mem == ROW_MAJOR) Dtranspose(C_T, C, M, N);

    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;

    local_t = csecond();
    C_gpu = Dvec_transfer_gpu(C_T, M * N);
    local_t = csecond() - local_t;
    ctrl->scatter_t += local_t;
  }

  gpu_timer_start(cuda_timer);
  stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M, N, K, &alpha, A_gpu, ldA,
                     B_gpu, ldB, &gpu_beta, C_gpu, M);

  gpu_timer_stop(cuda_timer);

  if (pred->Cadd_device_id == 0) cblas_dscal(M * N, beta, C, 1);

  cudaCheckErrors();
  ctrl->gpu_ex_t += gpu_timer_get(cuda_timer) / 1000;

  if (ctrl->C_mem == ROW_MAJOR) {
    local_t = csecond();
    hipMemcpy(C_T, C_gpu, M * N * sizeof(double), hipMemcpyDeviceToHost);
    local_t = csecond() - local_t;
    ctrl->gather_t += local_t;

    local_t = csecond();
    if (pred->Cadd_device_id == 0)
      Dtranspose_add(C, C_T, N, M);
    else
      Dtranspose(C, C_T, N, M);
    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;
  } else {
    local_t = csecond();
    if (pred->Cadd_device_id == 0) {
      local_t = csecond();
      hipMemcpy(C_T, C_gpu, M * N * sizeof(double), hipMemcpyDeviceToHost);
      local_t = csecond() - local_t;
      ctrl->gather_t += local_t;
      local_t = csecond();
      cblas_daxpy(N * M, 1.0, C_T, 1, C, 1);
      local_t = csecond() - local_t;
      ctrl->reduce_t += local_t;
    } else {
      local_t = csecond();
      hipMemcpy(C, C_gpu, M * N * sizeof(double), hipMemcpyDeviceToHost);
      local_t = csecond() - local_t;
      ctrl->gather_t += local_t;
    }
  }

  // FIXME: implement correct frees with switches
  gpu_free(A_gpu);
  gpu_free(B_gpu);
  gpu_free(C_gpu);
  pin_free(C_T);

  C_out = C;
  debug("<- gpu_dgemm()\n");
  return C_out;
}

double *hybrid_dgemm_Msplit(control_p ctrl, predict_p pred, size_t M, size_t N,
                            size_t K, double alpha, double *A, double *B,
                            double beta, double *C) {
  debug("-> hybrid_dgemm_Msplit()");
  if (ctrl->num_devices < 1)
    error(
        "hybrid_dgemm_Msplit -> 0 or less devices? What are you trying to "
        "do...");
  else if (ctrl->num_devices > 2)
    error(
        "hybrid_dgemm_Msplit -> Max 1 GPU + 1 CPU implemented (nice "
        "try,though).");

  if (pred->asynch_trans)
    error(
        "hybrid_dgemm_Msplit -> asynch transactions not implemented yet "
        "(nice try,though).");

  if (!pred->M_split || pred->M_split >= M)
    error("hybrid_dgemm_Msplit -> Full CPU/GPU versions do not belong here");
  if (!A) error("hybrid_dgemm_Msplit -> A is not malloc'ed correctly");
  if (!B) error("hybrid_dgemm_Msplit -> B is not malloc'ed correctly");
  if (beta != 0 && !C)
    error("hybrid_dgemm_Msplit -> C is not malloc'ed correctly");
  debug(
      "hybrid_dgemm_Msplit -> Trying your Matrix bounds (incomming "
      "segfaults)...");
  double test = A[M * K - 1];
  test = B[K * N - 1];
  if (beta || C) test = C[M * N - 1];
  debug("hybrid_dgemm_Msplit -> Passed.");
  double *C_out, local_t;

  double *A_cpu, *A_gpu, *B_cpu, *B_gpu, *C_cpu, *C_gpu;

  double *C_buffer, cpu_beta = 0, gpu_beta = 0;

  size_t M_gpu = pred->M_split, M_cpu = M - pred->M_split, ldA = 0, ldB = 0,
         ldC = 0, d_ldA = 0, d_ldB = 0, d_ldC = 0;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans

  CBLAS_LAYOUT cblas_target;
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);

  gpu_timer_p cuda_timer = gpu_timer_init();

  /// Setup A parts on host and device
  local_t = csecond();
  switch (ctrl->A_mem) {
    case (ROW_MAJOR):
      A_gpu = Dvec_transfer_gpu(A, M_gpu * K);
      gpu_op_A = HIPBLAS_OP_T;
      d_ldA = K;
      A_cpu = &(A[M_gpu * K]);
      ldA = K;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_A = CblasNoTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_A = CblasTrans;
      break;
    case (COL_MAJOR):
      gpu_op_A = HIPBLAS_OP_N;
      d_ldA = M_gpu;
      A_cpu = &(A[M_gpu]);
      ldA = M;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_A = CblasTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_A = CblasNoTrans;
      A_gpu = Dvec_chunk_transfer_gpu(A, K, M_gpu, M);
      break;
    default:
      error("hybrid_dgemm_Msplit -> Unreachable default reached ");
  }
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  /// Setup B parts on host and device
  local_t = csecond();
  B_cpu = B;
  B_gpu = Dvec_transfer_gpu(B, K * N);
  switch (ctrl->B_mem) {
    case (ROW_MAJOR):
      gpu_op_B = HIPBLAS_OP_T;
      d_ldB = N;
      ldB = N;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_B = CblasNoTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_B = CblasTrans;
      break;
    case (COL_MAJOR):
      gpu_op_B = HIPBLAS_OP_N;
      d_ldB = K;
      ldB = K;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_B = CblasTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_B = CblasNoTrans;
      break;
    default:
      error("hybrid_dgemm_Msplit -> Unreachable default reached ");
  }
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  /// Setup C parts on host and device
  d_ldC = M_gpu;

  if (!beta || !pred->Cadd_device_id) {
    cpu_beta = beta;
    local_t = csecond();
    C_gpu = (double *)gpu_malloc(M_gpu * N * sizeof(double));
    C_buffer = (double *)pin_malloc(M_gpu * N * sizeof(double));
    if (!C) C = (double *)malloc(M * N * sizeof(double));

    local_t = csecond() - local_t;
    ctrl->alloc_t += local_t;
  } else if (pred->Cadd_device_id == -1) {
    cpu_beta = gpu_beta = beta;
    if (ctrl->C_mem == ROW_MAJOR) {
      local_t = csecond();
      C_buffer = (double *)pin_malloc(M_gpu * N * sizeof(double));
      local_t = csecond() - local_t;
      ctrl->alloc_t += local_t;
      local_t = csecond();
      Dtranspose(C_buffer, C, M_gpu, N);
      local_t = csecond() - local_t;
      ctrl->transpose_t += local_t;
      local_t = csecond();
      C_gpu = Dvec_transfer_gpu(C_buffer, M_gpu * N);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
    } else if (ctrl->C_mem == COL_MAJOR) {
      local_t = csecond();
      C_buffer = (double *)pin_malloc(1 * sizeof(double));  /// Dummy for free
      C_gpu = Dvec_chunk_transfer_gpu(C, N, M_gpu, M);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
    }

  } else if (pred->Cadd_device_id == 1) {
    error("hybrid_dgemm_Msplit -> pred->Cadd_device_id == 1 Unimplemented.");
  }

  if (ctrl->C_mem == ROW_MAJOR) {
    cblas_target = CblasRowMajor;
    C_cpu = &(C[M_gpu * N]);
    ldC = N;
  } else if (ctrl->C_mem == COL_MAJOR) {
    C_cpu = &(C[M_gpu]);
    ldC = M;
    cblas_target = CblasColMajor;
  }

  if (!ldA || !ldB || !ldC || !d_ldA || !d_ldB || !d_ldC)
    error("hybrid_dgemm_Msplit -> Some ld_dim were not defined correctly (=0)");

  gpu_timer_start(cuda_timer);

  stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M_gpu, N, K, &alpha, A_gpu,
                     d_ldA, B_gpu, d_ldB, &gpu_beta, C_gpu, d_ldC);
  gpu_timer_stop(cuda_timer);

  if (pred->Cadd_device_id == 0) {
    local_t = csecond();
    switch (ctrl->C_mem) {
      case (ROW_MAJOR):
        cblas_dscal(N * M_gpu, beta, C, 1);
        break;
      case (COL_MAJOR):
        for (int i = 0; i < N; i++)
          cblas_daxpy(M_gpu, beta, &C[i * M], 1, &C_buffer[i * M_gpu], 1);
        break;
      default:
        error("hybrid_dgemm_Msplit -> A Unknown mem type");
    }
    local_t = csecond() - local_t;
    ctrl->reduce_t += local_t;
  }

  local_t = csecond();
  cblas_dgemm(cblas_target, cpu_op_A, cpu_op_B, M_cpu, N, K, alpha, A_cpu, ldA,
              B_cpu, ldB, cpu_beta, C_cpu, ldC);
  local_t = csecond() - local_t;
  ctrl->cpu_ex_t += local_t;
  cudaCheckErrors();

  ctrl->gpu_ex_t += (double)gpu_timer_get(cuda_timer) / 1000;

  if (ctrl->C_mem == ROW_MAJOR) {
    local_t = csecond();
    hipMemcpy(C_buffer, C_gpu, M_gpu * N * sizeof(double),
               hipMemcpyDeviceToHost);
    local_t = csecond() - local_t;
    ctrl->gather_t += local_t;
    local_t = csecond();
    if (pred->Cadd_device_id == 0)
      Dtranspose_add(C, C_buffer, N, M_gpu);
    else
      Dtranspose(C, C_buffer, N, M_gpu);
    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;

  } else if (ctrl->C_mem == COL_MAJOR) {
    local_t = csecond();
    Dvec_chunk_copy_from_gpu(C, C_gpu, N, M_gpu, M);

    if (pred->Cadd_device_id == 0)
      for (int i = 0; i < N; i++)
        cblas_daxpy(M_gpu, 1.0, &C_buffer[i * M_gpu], 1, &C[i * M], 1);
    local_t = csecond() - local_t;
    ctrl->gather_t += local_t;
  }

  // FIXME: implement correct frees with switches
  gpu_free(A_gpu);
  gpu_free(B_gpu);
  gpu_free(C_gpu);
  pin_free(C_buffer);

  C_out = C;
  debug("<- hybrid_dgemm_Msplit()\n");
  return C_out;
}

double *hybrid_dgemm_Nsplit(control_p ctrl, predict_p pred, size_t M, size_t N,
                            size_t K, double alpha, double *A, double *B,
                            double beta, double *C) {
  debug("-> hybrid_dgemm_Nsplit()");
  if (ctrl->num_devices < 1)
    error(
        "hybrid_dgemm_Nsplit -> 0 or less devices? What are you trying to "
        "do...");
  else if (ctrl->num_devices > 2)
    error(
        "hybrid_dgemm_Nsplit -> Max 1 GPU + 1 CPU implemented (nice "
        "try,though).");

  if (pred->asynch_trans)
    error(
        "hybrid_dgemm_Nsplit -> asynch transactions not implemented yet "
        "(nice try,though).");

  if (!pred->N_split || pred->N_split >= N)
    error("hybrid_dgemm_Nsplit -> Full CPU/GPU versions do not belong here");
  if (!A) error("hybrid_dgemm_Nsplit -> A is not malloc'ed correctly");
  if (!B) error("hybrid_dgemm_Nsplit -> B is not malloc'ed correctly");
  if (beta != 0 && !C)
    error("hybrid_dgemm_Nsplit -> C is not malloc'ed correctly");
  debug(
      "hybrid_dgemm_Nsplit -> Trying your Matrix bounds (incomming "
      "segfaults)...");
  double test = A[M * K - 1];
  test = B[K * N - 1];
  if (beta || C) test = C[M * N - 1];
  debug("hybrid_dgemm_Nsplit -> Passed.");
  double *C_out, local_t;

  double *A_cpu, *A_gpu, *B_cpu, *B_gpu, *C_cpu, *C_gpu;

  double *C_buffer, cpu_beta = 0, gpu_beta = 0;

  size_t N_gpu = pred->N_split, N_cpu = N - pred->N_split, ldA = 0, ldB = 0,
         ldC = 0, d_ldA = 0, d_ldB = 0, d_ldC = 0;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans

  CBLAS_LAYOUT cblas_target;
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);

  gpu_timer_p cuda_timer = gpu_timer_init();

  /// Setup A parts on host and device
  local_t = csecond();
  A_cpu = A;
  A_gpu = Dvec_transfer_gpu(A, M * K);

  switch (ctrl->A_mem) {
    case (ROW_MAJOR):
      gpu_op_A = HIPBLAS_OP_T;
      d_ldA = K;
      ldA = K;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_A = CblasNoTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_A = CblasTrans;
      break;
    case (COL_MAJOR):
      gpu_op_A = HIPBLAS_OP_N;
      d_ldA = M;
      ldA = M;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_A = CblasTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_A = CblasNoTrans;
      break;
    default:
      error("hybrid_dgemm_Nsplit -> Unreachable default reached ");
  }
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  /// Setup B parts on host and device
  local_t = csecond();
  switch (ctrl->B_mem) {
    case (ROW_MAJOR):
      gpu_op_B = HIPBLAS_OP_T;
      d_ldB = N_gpu;
      B_cpu = &(B[N_gpu]);
      ldB = N;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_B = CblasNoTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_B = CblasTrans;
      B_gpu = Dvec_chunk_transfer_gpu(B, K, N_gpu, N);
      break;
    case (COL_MAJOR):
      B_gpu = Dvec_transfer_gpu(B, N_gpu * K);
      gpu_op_B = HIPBLAS_OP_N;
      d_ldB = K;
      B_cpu = &(B[N_gpu * K]);
      ldB = K;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_B = CblasTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_B = CblasNoTrans;
      break;
    default:
      error("hybrid_dgemm_Nsplit -> Unreachable default reached ");
  }
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  /// Setup C parts on host and device
  d_ldC = M;

  if (!beta || !pred->Cadd_device_id) {
    cpu_beta = beta;
    local_t = csecond();
    C_gpu = (double *)gpu_malloc(M * N_gpu * sizeof(double));
    C_buffer = (double *)pin_malloc(M * N_gpu * sizeof(double));
    if (!C) C = (double *)malloc(M * N * sizeof(double));

    local_t = csecond() - local_t;
    ctrl->alloc_t += local_t;
  } else if (pred->Cadd_device_id == -1) {
    cpu_beta = gpu_beta = beta;
    if (ctrl->C_mem == ROW_MAJOR) {
      local_t = csecond();
      C_buffer = (double *)pin_malloc(M * N_gpu * sizeof(double));
      local_t = csecond() - local_t;
      ctrl->alloc_t += local_t;
      local_t = csecond();
      Dtranspose_stride_src(C_buffer, C, M, N_gpu, N);
      local_t = csecond() - local_t;
      ctrl->transpose_t += local_t;
      local_t = csecond();
      C_gpu = Dvec_transfer_gpu(C_buffer, M * N_gpu);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
    } else if (ctrl->C_mem == COL_MAJOR) {
      local_t = csecond();
      C_buffer = (double *)pin_malloc(1 * sizeof(double));  /// Dummy for free;
      C_gpu = Dvec_transfer_gpu(C, N_gpu * M);
      local_t = csecond() - local_t;
      ctrl->scatter_t += local_t;
    }

  } else if (pred->Cadd_device_id == 1) {
    error("hybrid_dgemm_Nsplit -> pred->Cadd_device_id == 1 Unimplemented.");
  }

  if (ctrl->C_mem == ROW_MAJOR) {
    cblas_target = CblasRowMajor;
    C_cpu = &(C[N_gpu]);
    ldC = N;
  } else if (ctrl->C_mem == COL_MAJOR) {
    C_cpu = &(C[N_gpu * M]);
    ldC = M;
    cblas_target = CblasColMajor;
  }

  if (!ldA || !ldB || !ldC || !d_ldA || !d_ldB || !d_ldC)
    error("hybrid_dgemm_Nsplit -> Some ld_dim were not defined correctly (=0)");

  gpu_timer_start(cuda_timer);

  stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M, N_gpu, K, &alpha, A_gpu,
                     d_ldA, B_gpu, d_ldB, &gpu_beta, C_gpu, d_ldC);
  gpu_timer_stop(cuda_timer);

  if (pred->Cadd_device_id == 0) {
    local_t = csecond();
    switch (ctrl->C_mem) {
      case (ROW_MAJOR):
        for (int i = 0; i < M; i++) cblas_dscal(N_gpu, beta, &C[i * N], 1);
        break;
      case (COL_MAJOR):
        cblas_dscal(N_gpu * M, beta, C, 1);
        break;
      default:
        error("hybrid_dgemm_Nsplit -> A Unknown mem type");
    }
    local_t = csecond() - local_t;
    ctrl->reduce_t += local_t;
  }

  local_t = csecond();
  cblas_dgemm(cblas_target, cpu_op_A, cpu_op_B, M, N_cpu, K, alpha, A_cpu, ldA,
              B_cpu, ldB, cpu_beta, C_cpu, ldC);
  local_t = csecond() - local_t;
  ctrl->cpu_ex_t += local_t;
  cudaCheckErrors();

  ctrl->gpu_ex_t += (double)gpu_timer_get(cuda_timer) / 1000;

  if (ctrl->C_mem == ROW_MAJOR) {
    local_t = csecond();
    hipMemcpy(C_buffer, C_gpu, M * N_gpu * sizeof(double),
               hipMemcpyDeviceToHost);
    local_t = csecond() - local_t;
    ctrl->gather_t += local_t;
    local_t = csecond();
    if (pred->Cadd_device_id == 0)
      Dtranspose_stride_dest_add(C, C_buffer, N_gpu, M, N);
    else
      Dtranspose_stride_dest(C, C_buffer, N_gpu, M, N);
    local_t = csecond() - local_t;
    ctrl->transpose_t += local_t;

  } else if (ctrl->C_mem == COL_MAJOR) {
    local_t = csecond();
    if (pred->Cadd_device_id == 0) {
      hipMemcpy(C_buffer, C_gpu, M * N_gpu * sizeof(double),
                 hipMemcpyDeviceToHost);
      cblas_daxpy(M * N_gpu, 1.0, C_buffer, 1, C, 1);
    } else
      hipMemcpy(C, C_gpu, M * N_gpu * sizeof(double), hipMemcpyDeviceToHost);

    local_t = csecond() - local_t;
    ctrl->gather_t += local_t;
  }

  // FIXME: implement correct frees with switches
  gpu_free(A_gpu);
  gpu_free(B_gpu);
  gpu_free(C_gpu);
  pin_free(C_buffer);

  C_out = C;
  debug("<- hybrid_dgemm_Nsplit()\n");
  return C_out;
}

double *hybrid_dgemm_Ksplit(control_p ctrl, predict_p pred, size_t M, size_t N,
                            size_t K, double alpha, double *A, double *B,
                            double beta, double *C) {
  debug("-> hybrid_dgemm_Ksplit()");
  if (ctrl->num_devices < 1)
    error(
        "hybrid_dgemm_Ksplit -> 0 or less devices? What are you trying to "
        "do...");
  else if (ctrl->num_devices > 2)
    error(
        "hybrid_dgemm_Ksplit -> Max 1 GPU + 1 CPU implemented (nice "
        "try,though).");

  if (pred->asynch_trans)
    error(
        "hybrid_dgemm_Ksplit -> asynch transactions not implemented yet "
        "(nice try,though).");

  if (!pred->K_split || pred->K_split >= K)
    error("hybrid_dgemm_Ksplit -> Full CPU/GPU versions do not belong here");
  if (!A) error("hybrid_dgemm_Ksplit -> A is not malloc'ed correctly");
  if (!B) error("hybrid_dgemm_Ksplit -> B is not malloc'ed correctly");
  if (beta != 0 && !C)
    error("hybrid_dgemm_Ksplit -> C is not malloc'ed correctly");
  debug(
      "hybrid_dgemm_Ksplit -> Trying your Matrix bounds (incomming "
      "segfaults)...");
  double test = A[M * K - 1];
  test = B[K * N - 1];
  if (beta || C) test = C[M * N - 1];
  debug("hybrid_dgemm_Ksplit -> Passed.");
  double *C_out, local_t;

  double *A_cpu, *A_gpu, *B_cpu, *B_gpu, *C_cpu, *C_gpu;

  double *C_buffer, cpu_beta = 0, gpu_beta = 0;

  size_t K_gpu = pred->K_split, K_cpu = K - pred->K_split, ldA = 0, ldB = 0,
         ldC = 0, d_ldA = 0, d_ldB = 0, d_ldC = 0;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans

  CBLAS_LAYOUT cblas_target;
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);

  gpu_timer_p cuda_timer = gpu_timer_init();

  /// Setup A parts on host and device
  local_t = csecond();
  switch (ctrl->A_mem) {
    case (ROW_MAJOR):
      gpu_op_A = HIPBLAS_OP_T;
      d_ldA = K_gpu;
      A_cpu = &(A[K_gpu]);
      ldA = K;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_A = CblasNoTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_A = CblasTrans;
      A_gpu = Dvec_chunk_transfer_gpu(A, M, K_gpu, K);
      break;
    case (COL_MAJOR):
      A_gpu = Dvec_transfer_gpu(A, K_gpu * M);
      gpu_op_A = HIPBLAS_OP_N;
      d_ldA = M;
      A_cpu = &(A[K_gpu * M]);
      ldA = M;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_A = CblasTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_A = CblasNoTrans;
      break;
    default:
      error("hybrid_dgemm_Ksplit -> Unreachable default reached ");
  }
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  /// Setup B parts on host and device
  local_t = csecond();
  switch (ctrl->B_mem) {
    case (ROW_MAJOR):
      B_gpu = Dvec_transfer_gpu(B, K_gpu * N);
      gpu_op_B = HIPBLAS_OP_T;
      d_ldB = N;
      B_cpu = &(B[K_gpu * N]);
      ldB = N;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_B = CblasNoTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_B = CblasTrans;
      break;
    case (COL_MAJOR):
      gpu_op_B = HIPBLAS_OP_N;
      d_ldB = K_gpu;
      B_cpu = &(B[K_gpu]);
      ldB = K;
      if (ctrl->C_mem == ROW_MAJOR)
        cpu_op_B = CblasTrans;
      else if (ctrl->C_mem == COL_MAJOR)
        cpu_op_B = CblasNoTrans;
      B_gpu = Dvec_chunk_transfer_gpu(B, N, K_gpu, K);
      break;
    default:
      error("hybrid_dgemm_Ksplit -> Unreachable default reached ");
  }
  local_t = csecond() - local_t;
  ctrl->scatter_t += local_t;

  /// Setup C parts on host and device
  local_t = csecond();
  d_ldC = M;
  C_buffer = (double *)pin_malloc(M * N * sizeof(double));

  if (!beta || !pred->Cadd_device_id || pred->Cadd_device_id == -1) {
    cpu_beta = beta;
    C_gpu = (double *)gpu_malloc(M * N * sizeof(double));
    if (!C) C = (double *)malloc(M * N * sizeof(double));
  } else if (pred->Cadd_device_id == 1) {
    error("hybrid_dgemm_Ksplit -> pred->Cadd_device_id == 1 Unimplemented.");
  }

  C_cpu = C;
  if (ctrl->C_mem == ROW_MAJOR) {
    cblas_target = CblasRowMajor;
    ldC = N;
  } else if (ctrl->C_mem == COL_MAJOR) {
    ldC = M;
    cblas_target = CblasColMajor;
  }
  local_t = csecond() - local_t;
  ctrl->alloc_t += local_t;

  if (!ldA || !ldB || !ldC || !d_ldA || !d_ldB || !d_ldC)
    error("hybrid_dgemm_Ksplit -> Some ld_dim were not defined correctly (=0)");

  gpu_timer_start(cuda_timer);

  stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M, N, K_gpu, &alpha, A_gpu,
                     d_ldA, B_gpu, d_ldB, &gpu_beta, C_gpu, d_ldC);
  gpu_timer_stop(cuda_timer);

  local_t = csecond();
  cblas_dgemm(cblas_target, cpu_op_A, cpu_op_B, M, N, K_cpu, alpha, A_cpu, ldA,
              B_cpu, ldB, cpu_beta, C_cpu, ldC);  /// ERROR!!!
  local_t = csecond() - local_t;
  ctrl->cpu_ex_t += local_t;
  cudaCheckErrors();

  ctrl->gpu_ex_t += (double)gpu_timer_get(cuda_timer) / 1000;

  local_t = csecond();
  hipMemcpy(C_buffer, C_gpu, M * N * sizeof(double), hipMemcpyDeviceToHost);
  local_t = csecond() - local_t;
  ctrl->gather_t += local_t;

  local_t = csecond();
  if (ctrl->C_mem == ROW_MAJOR)
    Dtranspose_add(C, C_buffer, N, M);
  else if (ctrl->C_mem == COL_MAJOR)
    cblas_daxpy(M * N, 1.0, C_buffer, 1, C, 1);
  local_t = csecond() - local_t;
  ctrl->reduce_t += local_t;

  // FIXME: implement correct frees with switches
  gpu_free(A_gpu);
  gpu_free(B_gpu);
  gpu_free(C_gpu);
  pin_free(C_buffer);

  C_out = C;
  debug("<- hybrid_dgemm_Ksplit()\n");
  return C_out;
}

int main(const int argc, const char *argv[]) {
  // print_devices();

  /*
    double *test, *test_T;

    test = Dvec_init_pinned(25, 42);
    test_T = Dvec_init_pinned(25, 0);

    for (int i = 0; i <5; i++){
            for (int j = 0; j <5; j++) fprintf(stderr,"%0.3lf ", test[5*i +j]);
            fprintf(stderr,"\n");
    }
    fprintf(stderr,"\n");
    fprintf(stderr,"\n");

    int s1 = 2, s2 = 5;
    Dtranspose(test_T, test, s1, s2);

    for (int i = 0; i <s1; i++){
            for (int j = 0; j <s2; j++) fprintf(stderr,"%0.3lf ", test[s2*i
    +j]); fprintf(stderr,"\n");
    }
    fprintf(stderr,"\n");
    for (int i = 0; i <s1*s2; i++)fprintf(stderr,"%0.3lf ", test_T[i]);
    fprintf(stderr,"\n");


    exit(1);
  */

  // Arguments: ( M N K A_mem{0,1} B_mem{0,1} C_mem{0,1} alpha) (beta
  // add_device) M_split N_split K_split cblas_mem{0,1}

  double alpha, beta;

  size_t M, N, K, itterations = 1;

  predict_p main_pred = (predict_p)malloc(sizeof(struct pred_str));
  control_p main_ctrl = (control_p)malloc(sizeof(struct control_str));

  main_ctrl->num_devices = 2;

  M = 100;
  K = 200;
  N = 300;
  main_ctrl->A_mem = ROW_MAJOR;
  main_ctrl->B_mem = ROW_MAJOR;
  main_ctrl->C_mem = ROW_MAJOR;
  alpha = 1.1;
  beta = 0;
  main_pred->Cadd_device_id = -1;
  main_pred->asynch_trans = 0;

  int ctr = 1;
  switch (argc) {
    case (14):
      M = atoi(argv[ctr++]);
      N = atoi(argv[ctr++]);
      K = atoi(argv[ctr++]);
      if (atoi(argv[ctr++]))
        main_ctrl->A_mem = COL_MAJOR;
      else
        main_ctrl->A_mem = ROW_MAJOR;
      if (atoi(argv[ctr++]))
        main_ctrl->B_mem = COL_MAJOR;
      else
        main_ctrl->B_mem = ROW_MAJOR;
      if (atoi(argv[ctr++]))
        main_ctrl->C_mem = COL_MAJOR;
      else
        main_ctrl->C_mem = ROW_MAJOR;
      alpha = atof(argv[ctr++]);
    case (7):
      beta = atof(argv[ctr++]);
      main_pred->Cadd_device_id = atoi(argv[ctr++]);
    case (5):
      main_pred->M_split = atoi(argv[ctr++]);
      main_pred->N_split = atoi(argv[ctr++]);
      main_pred->K_split = atoi(argv[ctr++]);
      break;
    default:
      error("Incorrect input arguments");
  }

  main_ctrl->alloc_t = main_ctrl->scatter_t = main_ctrl->transpose_t =
      main_ctrl->cpu_ex_t = main_ctrl->gpu_ex_t = main_ctrl->gather_t =
          main_ctrl->reduce_t = 0;

  // hipStream_t stream1, stream2;
  // hipStreamCreate(&stream1);
  // hipStreamCreate (&stream2);

  double *A, *B, *C, *C_comp, *C_buffer, *d_A, *d_B, *d_C, *C_T;
  size_t ldA, ldB, ldC;
  hipblasOperation_t gpu_op_A, gpu_op_B;  // HIPBLAS_OP_N, HIPBLAS_OP_T
  CBLAS_TRANSPOSE cpu_op_A, cpu_op_B;    // CblasNoTrans, CblasTrans
  CBLAS_LAYOUT cblas_layout;

  if (main_ctrl->A_mem == COL_MAJOR)
    ldA = M;
  else
    ldA = K;
  if (main_ctrl->B_mem == COL_MAJOR)
    ldB = K;
  else
    ldB = N;

  if (main_ctrl->C_mem == COL_MAJOR) {
    cblas_layout = CblasColMajor;
    ldC = M;
    if (main_ctrl->A_mem == COL_MAJOR) {
      cpu_op_A = CblasNoTrans;
      gpu_op_A = HIPBLAS_OP_N;
    } else {
      cpu_op_A = CblasTrans;
      gpu_op_A = HIPBLAS_OP_T;
    }
    if (main_ctrl->B_mem == COL_MAJOR) {
      cpu_op_B = CblasNoTrans;
      gpu_op_B = HIPBLAS_OP_N;
    } else {
      cpu_op_B = CblasTrans;
      gpu_op_B = HIPBLAS_OP_T;
    }
  } else {
    cblas_layout = CblasRowMajor;
    ldC = N;
    if (main_ctrl->A_mem == ROW_MAJOR) {
      cpu_op_A = CblasNoTrans;
      gpu_op_A = HIPBLAS_OP_T;
    } else {
      cpu_op_A = CblasTrans;
      gpu_op_A = HIPBLAS_OP_N;
    }
    if (main_ctrl->B_mem == ROW_MAJOR) {
      cpu_op_B = CblasNoTrans;
      gpu_op_B = HIPBLAS_OP_T;
    } else {
      cpu_op_B = CblasTrans;
      gpu_op_B = HIPBLAS_OP_N;
    }
  }

  int devices = 0;
  hipGetDeviceCount(&devices);
  if (main_pred->M_split + main_pred->N_split + main_pred->K_split &&
      devices < 1)
    error("Trying to execute something CUDA-related on node without CUDA GPUs");
  else
    hipSetDevice(devices - 1);

  double transpose_timer, cpu_timer = csecond(), total_t;
  float gpu_preproc_t = 0, gpu_comp_t = 0, gpu_reduce_t = 0;
  gpu_timer_p cuda_timer = gpu_timer_init();

  A = Dvec_init_pinned(M * K, 42);
  B = Dvec_init_pinned(K * N, 42);
  C = Dvec_init_pinned(M * N, 42);
  C_comp = Dvec_init_host(M * N, 0);
  Dvec_copy(C_comp, C, M * N);

  if (!strcmp(argv[ctr], "DEBUG")) {
    fprintf(stderr,
            "\nMatrix details: A(%s) B(%s) C(%s) -> M = %d, N = %d, K = %d\n",
            print_mem(main_ctrl->A_mem), print_mem(main_ctrl->B_mem),
            print_mem(main_ctrl->C_mem), M, N, K);
    fprintf(stderr, "Constants: alpha = %lf, beta = %lf\n", alpha, beta);
    fprintf(stderr,
            "Predicted values: M_split = %d, N_split = %d, K_split = %d "
            "Cadd_device "
            "= %d\n\n",
            main_pred->M_split, main_pred->N_split, main_pred->K_split,
            main_pred->Cadd_device_id);

    C_buffer = Dvec_init_host(M * N, 0);
    Dvec_copy(C_buffer, C, M * N);

    cpu_timer = csecond() - cpu_timer;
    fprintf(stderr,
            "Initializing Arrays on host (M=%d, N=%d, K=%d) t_init = %lf ms\n",
            M, N, K, cpu_timer * 1000);
    cpu_timer = csecond();
    for (int i = 0; i < 10; i++)
      cblas_dgemm(cblas_layout, cpu_op_A, cpu_op_B, M, N, K, alpha, A, ldA, B,
                  ldB, beta, C_comp, ldC);
    cpu_timer = csecond() - cpu_timer;
    fprintf(stderr, "MKL Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
    report_results(cpu_timer / 10, (long)M * K * (2 * N + 1),
                   (long)(M * K + K * N + M * N * 2) * sizeof(double));
    fprintf(stderr, "\n");

    transpose_timer = csecond();
    if (main_ctrl->C_mem == ROW_MAJOR) {
      C_T = (double *)pin_malloc(M * N * sizeof(double));
      Dtranspose(C_T, C, M, N);
    } else
      C_T = C;
    transpose_timer = csecond() - transpose_timer;

    gpu_timer_start(cuda_timer);
    d_A = Dvec_transfer_gpu(A, M * K);
    d_B = Dvec_transfer_gpu(B, K * N);
    d_C = Dvec_transfer_gpu(C_T, M * N);
    gpu_timer_stop(cuda_timer);
    gpu_preproc_t = gpu_timer_get(cuda_timer);

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    // hipblasSetStream(handle, stream1);
    gpu_timer_start(cuda_timer);

    for (int i = 0; i < 10; i++) {
      stat = hipblasDgemm(handle, gpu_op_A, gpu_op_B, M, N, K, &alpha, d_A, ldA,
                         d_B, ldB, &beta, d_C, M);
    }

    gpu_timer_stop(cuda_timer);
    cudaCheckErrors();
    gpu_comp_t = gpu_timer_get(cuda_timer);
    gpu_timer_start(cuda_timer);
    hipMemcpy(C_T, d_C, M * N * sizeof(double), hipMemcpyDeviceToHost);
    gpu_timer_stop(cuda_timer);
    transpose_timer = csecond() - transpose_timer;
    if (main_ctrl->C_mem == ROW_MAJOR) {
      Dtranspose(C, C_T, N, M);
      pin_free(C_T);
    } else
      C = C_T;
    transpose_timer = csecond() - transpose_timer;
    gpu_reduce_t = gpu_timer_get(cuda_timer);

    fprintf(stderr,
            "\nDevice overhead(M=%d, N=%d, K=%d) transpose = %lf ms, t_preproc "
            "= %lf "
            "ms, t_reduce = %lf ms\n",
            M, N, K, 1000 * transpose_timer, gpu_preproc_t, gpu_reduce_t);

    fprintf(stderr, "CUDA Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
    report_results((double)gpu_comp_t / 1000.0 / 10, (long)M * K * (2 * N + 1),
                   (long)(M * K + K * N + M * N * 2) *
                       sizeof(double));  //(M*N+(long)M*K*(3*N+1))
    fprintf(stderr, "\n");

    Dtest_equality(C_comp, C, M * N);
    fprintf(stderr, "\n");

    gpu_free(d_A);
    gpu_free(d_B);
    gpu_free(d_C);
    pin_free(C);

    C = Dvec_init_pinned(M * N, 0);
    Dvec_copy(C, C_buffer, M * N);
    Dvec_copy(C_comp, C_buffer, M * N);

    cblas_dgemm(cblas_layout, cpu_op_A, cpu_op_B, M, N, K, alpha, A, ldA, B,
                ldB, beta, C_comp, ldC);
  } else if (!strcmp(argv[ctr], "TEST"))
    cblas_dgemm(cblas_layout, cpu_op_A, cpu_op_B, M, N, K, alpha, A, ldA, B,
                ldB, beta, C_comp, ldC);
  else if (!strcmp(argv[ctr], "BENCHMARK")) {
    itterations = 1000;
    fprintf(stdout, "%d,%d,%d,%s,%s,%s,%lf,%lf,%d,%d,%d,%d", M, N, K,
            print_mem(main_ctrl->A_mem), print_mem(main_ctrl->B_mem),
            print_mem(main_ctrl->C_mem), alpha, beta, main_pred->Cadd_device_id,
            main_pred->M_split, main_pred->N_split, main_pred->K_split);
  } else
    error("Invalid run mode given");

  if (!main_pred->M_split + !main_pred->N_split + !main_pred->K_split < 2)
    error("split more than one dim for 2 devices.");

  if (main_pred->M_split == M || main_pred->N_split == N ||
      main_pred->K_split == K) {
    for (int it = 0; it < itterations; it++)
      C = gpu_dgemm(main_ctrl, main_pred, M, N, K, alpha, A, B, beta, C);

    total_t =
        (main_ctrl->gpu_ex_t + main_ctrl->scatter_t + main_ctrl->alloc_t +
         main_ctrl->transpose_t + main_ctrl->gather_t + main_ctrl->reduce_t) /
        itterations;

  } else if (main_pred->M_split) {
    for (int it = 0; it < itterations; it++)
      C = hybrid_dgemm_Msplit(main_ctrl, main_pred, M, N, K, alpha, A, B, beta,
                              C);

    fprintf(stderr, "Hybrid Sgemm(M_cpu=%d, N=%d, K=%d) CPU time = ",
            (M - main_pred->M_split), N, K);
    report_results(main_ctrl->cpu_ex_t / itterations,
                   (long)(M - main_pred->M_split) * K * (2 * N + 1),
                   (long)((M - main_pred->M_split) * K + K * N +
                          (M - main_pred->M_split) * N * 2) *
                       sizeof(double));  //(M*N+(long)M*K*(3*N+1))
    fprintf(stderr, "\n");

    fprintf(stderr, "Hybrid Sgemm(M_gpu=%d, N=%d, K=%d) GPU time = ",
            main_pred->M_split, N, K);
    report_results(
        main_ctrl->gpu_ex_t / itterations,
        (long)main_pred->M_split * K * (2 * N + 1),
        (long)(main_pred->M_split * K + K * N + main_pred->M_split * N * 2) *
            sizeof(double));  //(M*N+(long)M*K*(3*N+1))
    fprintf(stderr, "\n");

    total_t =
        (fmax(main_ctrl->cpu_ex_t, main_ctrl->gpu_ex_t) + main_ctrl->scatter_t +
         main_ctrl->alloc_t + main_ctrl->transpose_t + main_ctrl->gather_t +
         main_ctrl->reduce_t) /
        itterations;

  } else if (main_pred->N_split) {
    for (int it = 0; it < itterations; it++)
      C = hybrid_dgemm_Nsplit(main_ctrl, main_pred, M, N, K, alpha, A, B, beta,
                              C);

    fprintf(stderr, "Hybrid Sgemm(M=%d, N_cpu=%d, K=%d) CPU time = ", M,
            (N - main_pred->N_split), K);
    report_results(main_ctrl->cpu_ex_t / itterations,
                   (long)M * K * (2 * (N - main_pred->N_split) + 1),
                   (long)(M * K + K * (N - main_pred->N_split) +
                          M * (N - main_pred->N_split) * 2) *
                       sizeof(double));  //(M*N+(long)M*K*(3*N+1))
    fprintf(stderr, "\n");

    fprintf(stderr, "Hybrid Sgemm(M=%d, N_gpu=%d, K=%d) GPU time = ", M,
            main_pred->N_split, K);
    report_results(
        main_ctrl->gpu_ex_t / itterations,
        (long)M * K * (2 * main_pred->N_split + 1),
        (long)(M * K + K * main_pred->N_split + M * main_pred->N_split * 2) *
            sizeof(double));  //(M*N+(long)M*K*(3*N+1))
    fprintf(stderr, "\n");

    total_t =
        (fmax(main_ctrl->cpu_ex_t, main_ctrl->gpu_ex_t) + main_ctrl->scatter_t +
         main_ctrl->alloc_t + main_ctrl->transpose_t + main_ctrl->gather_t +
         main_ctrl->reduce_t) /
        itterations;

  } else if (main_pred->K_split) {
    for (int it = 0; it < itterations; it++)
      C = hybrid_dgemm_Ksplit(main_ctrl, main_pred, M, N, K, alpha, A, B, beta,
                              C);

    fprintf(stderr, "Hybrid Sgemm(M=%d, N=%d, K_cpu=%d) CPU time = ", M, N,
            K - main_pred->K_split);
    report_results(main_ctrl->cpu_ex_t / itterations,
                   (long)M * (K - main_pred->K_split) * (2 * N + 1),
                   (long)(M * (K - main_pred->K_split) +
                          (K - main_pred->K_split) * N + M * N * 2) *
                       sizeof(double));  //(M*N+(long)M*K*(3*N+1))
    fprintf(stderr, "\n");

    fprintf(stderr, "Hybrid Sgemm(M=%d, N=%d, K_gpu=%d) GPU time = ", M, N,
            main_pred->K_split);
    report_results(
        main_ctrl->gpu_ex_t / itterations,
        (long)M * main_pred->K_split * (2 * N + 1),
        (long)(M * main_pred->K_split + main_pred->K_split * N + M * N * 2) *
            sizeof(double));  //(M*N+(long)M*K*(3*N+1))
    fprintf(stderr, "\n");

    total_t =
        (fmax(main_ctrl->cpu_ex_t, main_ctrl->gpu_ex_t) + main_ctrl->scatter_t +
         main_ctrl->alloc_t + main_ctrl->transpose_t + main_ctrl->gather_t +
         main_ctrl->reduce_t) /
        itterations;

  } else {
    debug("Not spliting at all, execute the whole on host");

    cpu_timer = csecond();
    for (int it = 0; it < itterations; it++)
      cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M, N, K, alpha, A,
                  K, B, N, beta, C, N);
    total_t = (csecond() - cpu_timer) / itterations;
    fprintf(stderr, "MKL Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
    report_results(total_t, (long)M * K * (2 * N + 1),
                   (long)(M * K + K * N + M * N * 2) * sizeof(double));
    fprintf(stderr, "\n");
  }

  fprintf(stderr,
          "Device overhead(M=%d, N=%d, K=%d) pin_alloc = %lf ms, scatter = %lf "
          "ms, "
          "transpose = %lf ms, gather = %lf ms, reduce = %lf ms\n",
          M, N, K, 1000 * main_ctrl->alloc_t / itterations,
          1000 * main_ctrl->scatter_t / itterations,
          1000 * main_ctrl->transpose_t / itterations,
          1000 * main_ctrl->gather_t / itterations,
          1000 * main_ctrl->reduce_t / itterations);

  fprintf(stderr, "Total Sgemm(M=%d, N=%d, K=%d) ", M, N, K);
  report_results(total_t, (long)M * K * (2 * N + 1),
                 (long)(M * K + K * N + M * N * 2) *
                     sizeof(double));  //(M*N+(long)M*K*(3*N+1))
  fprintf(stderr, "\n");

  if (1 == itterations)
    Dtest_equality(C_comp, C, M * N);
  else
    fprintf(
        stdout, ",%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n",
        main_ctrl->alloc_t / itterations, main_ctrl->scatter_t / itterations,
        main_ctrl->transpose_t / itterations, main_ctrl->gather_t / itterations,
        main_ctrl->reduce_t / itterations, main_ctrl->cpu_ex_t / itterations,
        main_ctrl->gpu_ex_t / itterations, total_t);

  return 0;
}
